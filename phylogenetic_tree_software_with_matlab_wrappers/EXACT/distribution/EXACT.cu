#include "hip/hip_runtime.h"
// Code to produce a tree from an index using prufer codes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include "timerc.h"
#include <omp.h>
#include <sys/resource.h>
#include <sys/time.h>
#include <string.h>

// we probably will never consider trees with more than 13 nodes so we can use this number as a fixed bound which helps optimize the code
#define MAX_N_NODES 13
#define delta (1e6)
#define topk 100

// this allows us to change the precision with which we compute our results
typedef float realnumber;
typedef unsigned long int longint;
typedef short int shortint;



// this allows us to catch errors produced by the GPU
#define gerror(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}



// maybe we should change this to a simple array later.
typedef struct {
    shortint first;
    shortint second;
} edge;


//
//
//
//
// ------------------  PRIOR ON TREE COST
//
//      ---------------------- THIS MIGHT BE USED TO EXCLUDE SOME TOPOLOGIES THAT WE KNOW WILL NOT HAPPEN
//				----- IF WE WANT TO EXCLUDE SOME TOPOLOGY, WE CAN SIMPLY RETURN A VALUE OF 10000 OR ABOVE
//

__device__ __host__ realnumber inline tree_prior(shortint num_tree_vertices, shortint *adjacency_mat, shortint * adjacency_list, shortint * degrees){

	return 0;

}


//
//
//
//
// ------------------  COST TRANSFORMATION FUNCTION
//
//

__device__ __host__ realnumber inline cost_trans_function(realnumber x){

    return x;

}



//
//
//
// ***************************** START OF SORTING CODE ***********************************************************************************************************
//
//
// we need to do several sorting operations so here we have efficient code to do sorting

//This sorts smaller to larger
__device__ __host__ void insertion_sort(realnumber *a, shortint n)
{
    for (shortint i = 1; i < n; i++)
    {
        realnumber tmp = a[i];
        shortint j = i;
        for (; j && tmp < a[j - 1]; --j)
            a[j] = a[j - 1];
        a[j] = tmp;
    }
}

#define CMP_SWAP(i, j) if (a[i] > a[j])  { realnumber tmp = a[i]; a[i] = a[j]; a[j] = tmp; }

// for the sorting networks the following websites are useful    http://jgamble.ripco.net/cgi-bin/nw.cgi?inputs=7&algorithm=best&output=svg       and      http://stackoverflow.com/questions/4770651/what-is-the-fastest-sorting-algorithm-for-a-small-number-of-integers

__device__ __host__ void sort14_network(realnumber *a){
    CMP_SWAP(0,1);CMP_SWAP(2,3);CMP_SWAP(4,5);CMP_SWAP(6,7);CMP_SWAP(8,9);CMP_SWAP(10,11);CMP_SWAP(12,13);
    CMP_SWAP(0,2);CMP_SWAP(4,6);CMP_SWAP(8,10);CMP_SWAP(1,3);CMP_SWAP(5,7);CMP_SWAP(9,11);
    CMP_SWAP(0,4);CMP_SWAP(8,12);CMP_SWAP(1,5);CMP_SWAP(9,13);CMP_SWAP(2,6);CMP_SWAP(3,7);
    CMP_SWAP(0,8);CMP_SWAP(1,9);CMP_SWAP(2,10);CMP_SWAP(3,11);CMP_SWAP(4,12);CMP_SWAP(5,13);
    CMP_SWAP(5,10);CMP_SWAP(6,9);CMP_SWAP(3,12);CMP_SWAP(7,11);CMP_SWAP(1,2);CMP_SWAP(4,8);
    CMP_SWAP(1,4);CMP_SWAP(7,13);CMP_SWAP(2,8);CMP_SWAP(5,6);CMP_SWAP(9,10);
    CMP_SWAP(2,4);CMP_SWAP(11,13);CMP_SWAP(3,8);CMP_SWAP(7,12);
    CMP_SWAP(6,8);CMP_SWAP(10,12);CMP_SWAP(3,5);CMP_SWAP(7,9);
    CMP_SWAP(3,4);CMP_SWAP(5,6);CMP_SWAP(7,8);CMP_SWAP(9,10);CMP_SWAP(11,12);
    CMP_SWAP(6,7);CMP_SWAP(8,9);
}


__device__ __host__ void sort13_network(realnumber *a){
    CMP_SWAP(1,7);CMP_SWAP(9,11);CMP_SWAP(3,4);CMP_SWAP(5,8);CMP_SWAP(0,12);CMP_SWAP(2,6);
    CMP_SWAP(0,1);CMP_SWAP(2,3);CMP_SWAP(4,6);CMP_SWAP(8,11);CMP_SWAP(7,12);CMP_SWAP(5,9);
    CMP_SWAP(0,2);CMP_SWAP(3,7);CMP_SWAP(10,11);CMP_SWAP(1,4);CMP_SWAP(6,12);
    CMP_SWAP(7,8);CMP_SWAP(11,12);CMP_SWAP(4,9);CMP_SWAP(6,10);
    CMP_SWAP(3,4);CMP_SWAP(5,6);CMP_SWAP(8,9);CMP_SWAP(10,11);CMP_SWAP(1,7);
    CMP_SWAP(2,6);CMP_SWAP(9,11);CMP_SWAP(1,3);CMP_SWAP(4,7);CMP_SWAP(8,10);CMP_SWAP(0,5);
    CMP_SWAP(2,5);CMP_SWAP(6,8);CMP_SWAP(9,10);
    CMP_SWAP(1,2);CMP_SWAP(3,5);CMP_SWAP(7,8);CMP_SWAP(4,6);
    CMP_SWAP(2,3);CMP_SWAP(4,5);CMP_SWAP(6,7);CMP_SWAP(8,9);
    CMP_SWAP(3,4);CMP_SWAP(5,6);
}

__device__ __host__ void sort12_network(realnumber *a){
    CMP_SWAP(0,1);CMP_SWAP(2,3);CMP_SWAP(4,5);CMP_SWAP(6,7);CMP_SWAP(8,9);CMP_SWAP(10,11);
    CMP_SWAP(1,3);CMP_SWAP(5,7);CMP_SWAP(9,11);CMP_SWAP(0,2);CMP_SWAP(4,6);CMP_SWAP(8,10);
    CMP_SWAP(1,2);CMP_SWAP(5,6);CMP_SWAP(9,10);CMP_SWAP(0,4);CMP_SWAP(7,11);
    CMP_SWAP(1,5);CMP_SWAP(6,10);CMP_SWAP(3,7);CMP_SWAP(4,8);
    CMP_SWAP(5,9);CMP_SWAP(2,6);CMP_SWAP(0,4);CMP_SWAP(7,11);CMP_SWAP(3,8);
    CMP_SWAP(1,5);CMP_SWAP(6,10);CMP_SWAP(2,3);CMP_SWAP(8,9);
    CMP_SWAP(1,4);CMP_SWAP(7,10);CMP_SWAP(3,5);CMP_SWAP(6,8);
    CMP_SWAP(2,4);CMP_SWAP(7,9);CMP_SWAP(5,6);
    CMP_SWAP(3,4);CMP_SWAP(7,8);
}


__device__ __host__ void sort11_network(realnumber *a){
    CMP_SWAP(0,1);CMP_SWAP(2,3);CMP_SWAP(4,5);CMP_SWAP(6,7);CMP_SWAP(8,9);
    CMP_SWAP(1,3);CMP_SWAP(5,7);CMP_SWAP(0,2);CMP_SWAP(4,6);CMP_SWAP(8,10);
    CMP_SWAP(1,2);CMP_SWAP(5,6);CMP_SWAP(9,10);CMP_SWAP(0,4);CMP_SWAP(3,7);
    CMP_SWAP(1,5);CMP_SWAP(6,10);CMP_SWAP(4,8);
    CMP_SWAP(5,9);CMP_SWAP(2,6);CMP_SWAP(0,4);CMP_SWAP(3,8);
    CMP_SWAP(1,5);CMP_SWAP(6,10);CMP_SWAP(2,3);CMP_SWAP(8,9);
    CMP_SWAP(1,4);CMP_SWAP(7,10);CMP_SWAP(3,5);CMP_SWAP(6,8);
    CMP_SWAP(2,4);CMP_SWAP(7,9);CMP_SWAP(5,6);
    CMP_SWAP(3,4);CMP_SWAP(7,8);
}


__device__ __host__ void sort10_network(realnumber *a){
    CMP_SWAP(4,9);CMP_SWAP(3,8);CMP_SWAP(2,7);CMP_SWAP(1,6);CMP_SWAP(0,5);
    CMP_SWAP(1,4);CMP_SWAP(6,9);CMP_SWAP(0,3);CMP_SWAP(5,8);
    CMP_SWAP(0,2);CMP_SWAP(3,6);CMP_SWAP(7,9);
    CMP_SWAP(0,1);CMP_SWAP(2,4);CMP_SWAP(5,7);CMP_SWAP(8,9);
    CMP_SWAP(1,2);CMP_SWAP(4,6);CMP_SWAP(7,8);CMP_SWAP(3,5);
    CMP_SWAP(2,5);CMP_SWAP(6,8);CMP_SWAP(1,3);CMP_SWAP(4,7);
    CMP_SWAP(2,3);CMP_SWAP(6,7);
    CMP_SWAP(3,4);CMP_SWAP(5,6);
    CMP_SWAP(4,5);
}

__device__ __host__ void sort9_network(realnumber *a){
    CMP_SWAP(0,1);CMP_SWAP(3,4);CMP_SWAP(6,7);
    CMP_SWAP(1,2);CMP_SWAP(4,5);CMP_SWAP(7,8);
    CMP_SWAP(0,1);CMP_SWAP(3,4);CMP_SWAP(6,7);CMP_SWAP(2,5);
    CMP_SWAP(0,3);CMP_SWAP(1,4);CMP_SWAP(5,8);
    CMP_SWAP(3,6);CMP_SWAP(4,7);CMP_SWAP(2,5);
    CMP_SWAP(0,3);CMP_SWAP(1,4);CMP_SWAP(5,7);CMP_SWAP(2,6);
    CMP_SWAP(1,3);CMP_SWAP(4,6);
    CMP_SWAP(2,4);CMP_SWAP(5,6);
    CMP_SWAP(2,3);
}

__device__ __host__ void sort8_network(realnumber *a){
    CMP_SWAP(0, 1); CMP_SWAP(2, 3); CMP_SWAP(4, 5); CMP_SWAP(6, 7);
    CMP_SWAP(0, 2); CMP_SWAP(1, 3); CMP_SWAP(4, 6); CMP_SWAP(5, 7);
    CMP_SWAP(1, 2); CMP_SWAP(5, 6); CMP_SWAP(0, 4); CMP_SWAP(3, 7);
    CMP_SWAP(1, 5); CMP_SWAP(2, 6);
    CMP_SWAP(1, 4); CMP_SWAP(3, 6);
    CMP_SWAP(2, 4); CMP_SWAP(3, 5);
    CMP_SWAP(3, 4);
}

__device__ __host__ void sort7_network(realnumber *a){
    CMP_SWAP(1,2);CMP_SWAP(3,4);CMP_SWAP(5,6);
    CMP_SWAP(0,2);CMP_SWAP(3,5);CMP_SWAP(4,6);
    CMP_SWAP(0,1);CMP_SWAP(4,5);CMP_SWAP(2,6);
    CMP_SWAP(0,4);CMP_SWAP(1,5);
    CMP_SWAP(0,3);CMP_SWAP(2,5);
    CMP_SWAP(1,3);CMP_SWAP(2,4);
    CMP_SWAP(2,3);
}

__device__ __host__ void sort6_network(realnumber *a){
    CMP_SWAP(1,2);CMP_SWAP(4,5);
    CMP_SWAP(0,2);CMP_SWAP(3,5);
    CMP_SWAP(0,1);CMP_SWAP(3,4);CMP_SWAP(2,5);
    CMP_SWAP(0,3);CMP_SWAP(1,4);
    CMP_SWAP(2,4);CMP_SWAP(1,3);
    CMP_SWAP(2,3);
}

__device__ __host__ void sort5_network(realnumber *a){
    CMP_SWAP(0,1);CMP_SWAP(3,4);
    CMP_SWAP(2,4);
    CMP_SWAP(2,3);CMP_SWAP(1,4);
    CMP_SWAP(0,3);
    CMP_SWAP(0,2);CMP_SWAP(1,3);
    CMP_SWAP(1,2);
}

__device__ __host__ void sort4_network(realnumber *a){
    CMP_SWAP(0,1);CMP_SWAP(2,3);
    CMP_SWAP(0,2);CMP_SWAP(1,3);
    CMP_SWAP(1,2);
}

__device__ __host__ void sort3_network(realnumber *a){
    CMP_SWAP(1,2);
    CMP_SWAP(0,2);
    CMP_SWAP(0,1);
}

__device__ __host__ void sort2_network(realnumber *a){
    CMP_SWAP(0,1);
}


//This sorts smaller to larger
__device__ __host__ void sort_using_networks(realnumber *a, shortint n){
    
    // we only process if the number of elements is  between 2 and 14
    if (n==1 || n > 14){
        return;
    }
    
    switch(n) {
            
        case 2  :
            sort2_network(a);
            break;
            
        case 3  :
            sort3_network(a);
            break;
            
        case 4  :
            sort4_network(a);
            break;
            
        case 5  :
            sort5_network(a);
            break;
            
        case 6  :
            sort6_network(a);
            break;
            
        case 7  :
            sort7_network(a);
            break;
            
        case 8  :
            sort8_network(a);
            break;
            
        case 9  :
            sort9_network(a);
            break;
            
        case 10  :
            sort10_network(a);
            break;
            
        case 11  :
            sort11_network(a);
            break;
            
        case 12  :
            sort12_network(a);
            break;
            
        case 13  :
            sort13_network(a);
            break;
            
        case 14  :
            sort14_network(a);
            break;
    }
    
}



//
//
//
// ***************************** END OF SORTING CODE ***********************************************************************************************************
//
//




//
//
//
//****************************** START OF BEI'S CODE WITH ARRAYS NO MALLOCS NO STRUCTS ******************************
//
//
//
//
//

/*
 LIST OF ARRAYS ABOUT TREE:
 realnumber value[], size = 2*MAX_N_NODES + 1
 realnumber z[], size = 2*MAX_N_NODES + 1
 realnumber status[], size = 2*MAX_N_NODES + 1
 status: (0 : free), (1 : branch), (2 : fixed), (3 : effective)
 int father[], size = 2*MAX_N_NODES + 1
 int num_kids[], size = MAX_N_NODES + 1 ---> Use final_degrees[] instead
 int kids[], size = (MAX_N_NODES + 1) * (MAX_N_NODES + 1)
 */
/*
 QUEUE OPERATIONS:
 push_back: back = (back + 1) % capacity;
 ++size;
 pop_front: front = (front + 1) % capacity;
 --size;
 */
/*
 STACK OPERATIONS:
 push_back: back = (back + 1) % capacity;
 ++size;
 pop_back: --size;
 back = (back - 1) % capacity;
 */

// Some path finding functions:

/*
 This function finds the path from the node at start_ind to the node at node_ind, and store the path in holder (a stack)
 */
__host__ __device__ void up_to_node_ind(shortint start_ind, shortint node_ind, shortint status[], shortint level[], shortint father[], shortint num_kids[], shortint kids[], shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    if (start_ind == node_ind) {
        return;
    }
    shortint current_ind = father[start_ind];
    if (current_ind < 0) {
        return;
    }
    *holder_front = 0;
    *holder_back = 0;
    *holder_size = 1;
    holder[*holder_back] = current_ind;
    while (current_ind != node_ind) {
        current_ind = father[current_ind];
        if (current_ind < 0) {
            return;
        }
        *holder_back = (*holder_back + 1) % (MAX_N_NODES + 1);
        holder[*holder_back] = current_ind;
        ++(*holder_size);
    }
}

/*
 This function finds the path from the node at start_ind to the nearest branch node above it, and store the path in holder (a stack)
 */
__host__ __device__ void up_to_branch_ind(shortint start_ind, shortint status[], shortint level[], shortint father[], shortint num_kids[], shortint kids[], shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    shortint current_ind = father[start_ind];
    if (current_ind < 0) {
        return;
    }
    *holder_front = 0;
    *holder_back = 0;
    *holder_size = 1;
    holder[*holder_back] = current_ind;
    while (status[current_ind] != 1) {
        current_ind = father[current_ind];
        if (current_ind < 0) {
            return;
        }
        *holder_back = (*holder_back + 1) % (MAX_N_NODES + 1);
        holder[*holder_back] = current_ind;
        ++(*holder_size);
    }
}

/*
 This function uses BFS on the entire tree, find all fixed nodes and store them in fixed_queue[], and levels of all nodes and stored in level[]
 */
__host__ __device__ void bfs_get_level(shortint status[], shortint level[], shortint num_kids[], shortint kids[], shortint fixed_queue[], shortint *fixed_queue_front, shortint *fixed_queue_back, shortint *fixed_queue_size) {
    shortint queue[MAX_N_NODES + 1];
    shortint queue_front = 0;
    queue[queue_front] = 0;
    shortint queue_back = 0;
    shortint queue_size = 1;
    level[0] = 0;
    while (queue_size != 0) {
        shortint current_ind = queue[queue_front];
        queue_front = (queue_front + 1) % (MAX_N_NODES + 1);
        --queue_size;
        if (status[current_ind] == 2) {
            *fixed_queue_back = (*fixed_queue_back + 1) % (MAX_N_NODES + 1);
            fixed_queue[*fixed_queue_back] = current_ind;
            ++(*fixed_queue_size);
        }
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            // Note: this BFS will never look at effective nodes produced during compression
            queue_back = (queue_back + 1) % (MAX_N_NODES + 1);
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            queue[queue_back] = kid_ind;
            ++queue_size;
            level[kid_ind] = level[current_ind] + 1;
        }
    }
}

/*
 This function uses BFS to find nodes up to fixed nodes (not inlcuding effective nodes), sore found branch nodes in branch_stack[], and store found fixed nodes in fixed_stack[]:
 */
__host__ __device__ void bfs_array_to_fixed(shortint start_ind, shortint status[], shortint num_kids[], shortint kids[], shortint branch_stack[], shortint *branch_stack_back, shortint *branch_stack_size, shortint fixed_stack[], shortint *fixed_stack_back, shortint *fixed_stack_size) {
    shortint queue[MAX_N_NODES + 1];
    shortint queue_front = 0;
    queue[queue_front] = start_ind;
    shortint queue_back = 0;
    shortint queue_size = 1;
    while (queue_size != 0) {
        shortint current_ind = queue[queue_front];
        queue_front = (queue_front + 1) % (MAX_N_NODES + 1);
        --queue_size;
        if (status[current_ind] == 1) {
            *branch_stack_back = (*branch_stack_back + 1) % (MAX_N_NODES + 1);
            branch_stack[*branch_stack_back] = current_ind;
            ++(*branch_stack_size);
        }
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            // Note: this BFS will never look at effective nodes produced during compression
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            if (status[kid_ind] != 2 && status[kid_ind] != 3) {
                queue_back = (queue_back + 1) % (MAX_N_NODES + 1);
                queue[queue_back] = kid_ind;
                ++queue_size;
            }
            else {
                *fixed_stack_back = (*fixed_stack_back + 1) % (MAX_N_NODES + 1);
                fixed_stack[*fixed_stack_back] = kid_ind;
                ++(*fixed_stack_size);
            }
        }
    }
}

/*
 This function uses BFS to find nodes up to fixed nodes (inlcuding effective nodes), store found branch nodes in branch_stack[], and store found fixed nodes (and effective nodes) in fixed_stack[]:
 */
__host__ __device__ void bfs_array_to_fixed_effective(shortint start_ind, shortint status[], shortint num_kids[], shortint kids[], shortint branch_stack[], shortint *branch_stack_back, shortint *branch_stack_size, shortint fixed_stack[], shortint *fixed_stack_back, shortint *fixed_stack_size) {
    shortint queue[MAX_N_NODES + 1];
    shortint queue_front = 0;
    queue[queue_front] = start_ind;
    shortint queue_back = 0;
    shortint queue_size = 1;
    while (queue_size != 0) {
        shortint current_ind = queue[queue_front];
        queue_front = (queue_front + 1) % (MAX_N_NODES + 1);
        --queue_size;
        if (status[current_ind] == 1) {
            *branch_stack_back = (*branch_stack_back + 1) % (MAX_N_NODES + 1);
            branch_stack[*branch_stack_back] = current_ind;
            ++(*branch_stack_size);
        }
        if (num_kids[current_ind] > 0) {
            for (shortint i = 0; i <= num_kids[current_ind]; ++i) {
                // Note: this BFS will look at effective nodes produced during compression
                shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
                // If that last spot has no effective node in it:
                if (kid_ind == -2) {
                    break;
                }
                else if (status[kid_ind] != 2 && status[kid_ind] != 3) {
                    queue_back = (queue_back + 1) % (MAX_N_NODES + 1);
                    queue[queue_back] = kid_ind;
                    ++queue_size;
                }
                else {
                    // Keep record of ending fixed nodes (including effective nodes)
                    *fixed_stack_back = (*fixed_stack_back + 1) % (MAX_N_NODES + 1);
                    fixed_stack[*fixed_stack_back] = kid_ind;
                    ++(*fixed_stack_size);
                }
            }
        }
    }
}

/*
 This function uses BFS to find positions (stored in z[]) and speeds (stored in v[]) of unstretched nodes
 */
__host__ __device__ void bfs_array_unstretched(shortint status[], shortint level[], shortint father[], shortint num_kids[], shortint kids[], shortint seen[], realnumber z[], realnumber v[]) {
    shortint queue[MAX_N_NODES + 1];
    shortint queue_front = 0;
    queue[queue_front] = 0;
    shortint queue_back = 0;
    shortint queue_size = 1;
    while (queue_size != 0) {
        shortint current_ind = queue[queue_front];
        queue_front = (queue_front + 1) % (MAX_N_NODES + 1);
        --queue_size;
        if (seen[current_ind] == 0 && status[current_ind] != 2) {
            z[current_ind] = z[father[current_ind]];
            v[current_ind] = v[father[current_ind]];
            seen[current_ind] = 1;
        }
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            // Note: this BFS will never look at effective nodes produced during compression
            queue_back = (queue_back + 1) % (MAX_N_NODES + 1);
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            queue[queue_back] = kid_ind;
            ++queue_size;
        }
    }
}

/*
 This function check whether the node at node_ind is a branch node or not
 */
__host__ __device__ shortint is_branch_array(shortint node_ind, shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim) {
    if (status[node_ind] == 2) {
        return 0;
    }
    // Define a branch_stack;
    shortint branch_stack[MAX_N_NODES + 1];
    shortint branch_stack_back = MAX_N_NODES;
    shortint branch_stack_size = 0;
    // Define a fixed_stack;
    shortint fixed_stack[MAX_N_NODES + 1];
    shortint fixed_stack_back = MAX_N_NODES;
    shortint fixed_stack_size = 0;
    bfs_array_to_fixed(node_ind, status, num_kids, kids, branch_stack, &branch_stack_back, &branch_stack_size, fixed_stack, &fixed_stack_back, &fixed_stack_size);
    shortint count_fixed = 0;
    while (fixed_stack_size != 0) {
        shortint current_ind = fixed_stack[fixed_stack_back];
        fixed_stack_back = (fixed_stack_back - 1) % (MAX_N_NODES + 1);
        --(fixed_stack_size);
        if (current_ind > 0 && current_ind <= dim) {
            ++count_fixed;
        }
        if (count_fixed > 1) {
            return 1;
        }
    }
    return 0;
}

/*
 This function implements the compression procedure starting with a branching node at branch_ind, create and store the data of a new effective node (associated with this branch node) from compressing:
 */
__host__ __device__ void compress_array(shortint branch_ind, realnumber value[], realnumber z[], shortint status[], shortint level[], shortint father[], shortint num_kids[], shortint kids[], shortint seen_fixed[], shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size, shortint *eff_ind) {
    ++(*eff_ind);
    // Define a branch_stack;
    shortint branch_stack[MAX_N_NODES + 1];
    shortint branch_stack_back = MAX_N_NODES;
    shortint branch_stack_size = 0;
    // Define a fixed_stack;
    shortint fixed_stack[MAX_N_NODES + 1];
    shortint fixed_stack_back = MAX_N_NODES;
    shortint fixed_stack_size = 0;
    // Define a fixed_queue;
    shortint fixed_queue[MAX_N_NODES + 1];
    shortint fixed_queue_front = 0;
    shortint fixed_queue_back = MAX_N_NODES;
    shortint fixed_queue_size = 0;
    // Find other branch nodes and fixed nodes in effective tree:
    bfs_array_to_fixed_effective(branch_ind, status, num_kids, kids, branch_stack, &branch_stack_back, &branch_stack_size, fixed_stack, &fixed_stack_back, &fixed_stack_size);
    realnumber denominator = 0;
    realnumber numerator = 0;
    *holder_front = 0;
    *holder_back = MAX_N_NODES;
    *holder_size = 0;
    while (fixed_stack_size != 0) {
        shortint current_ind = fixed_stack[fixed_stack_back];
        fixed_stack_back = (fixed_stack_back - 1) % (MAX_N_NODES + 1);
        --(fixed_stack_size);
        fixed_queue_back = (fixed_queue_back + 1) % (MAX_N_NODES + 1);
        fixed_queue[fixed_queue_back] = current_ind;
        ++(fixed_queue_size);
        if (seen_fixed[current_ind] == 0 && status[current_ind] != 3) {
            denominator += 1.0 / (level[current_ind] - level[branch_ind]);
        }
        else if (seen_fixed[current_ind] == 0 && status[current_ind] == 3) {
            up_to_branch_ind(current_ind, status, level, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
            realnumber diff = level[holder[*holder_back]] - level[branch_ind];
            realnumber dist = (1.0 / value[current_ind]) + diff;
            denominator += 1.0 / dist;
        }
    }
    *holder_front = 0;
    *holder_back = MAX_N_NODES;
    *holder_size = 0;
    while (fixed_queue_size != 0) {
        shortint current_ind = fixed_queue[fixed_queue_front];
        fixed_queue_front = (fixed_queue_front + 1) % (MAX_N_NODES + 1);
        --(fixed_queue_size);
        if (seen_fixed[current_ind] == 0 && status[current_ind] != 3) {
            numerator += z[current_ind] / (level[current_ind] - level[branch_ind]);
            seen_fixed[current_ind] = 1;
        }
        else if (seen_fixed[current_ind] == 0 && status[current_ind] == 3) {
            up_to_branch_ind(current_ind, status, level, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
            realnumber diff = level[holder[*holder_back]] - level[branch_ind];
            realnumber dist = (1.0 / value[current_ind]) + diff;
            numerator += z[current_ind] / dist;
            seen_fixed[current_ind] = 1;
        }
    }
    // Create a new effective node from compressing:
    status[*eff_ind] = 3;
    value[*eff_ind] = denominator;
    z[*eff_ind] = numerator / denominator;
    father[*eff_ind] = branch_ind;
}

/*
 This function updates positions z[] AND speeds v[] at a given t:
 */
__host__ __device__ void update_z_array(realnumber t, realnumber v[], shortint fixed_queue[], shortint *fixed_queue_front, shortint *fixed_queue_back, shortint *fixed_queue_size, realnumber value[], realnumber z[], shortint status[], shortint level[], shortint father[], shortint num_kids[], shortint kids[], shortint dim) {
    shortint seen[MAX_N_NODES + 1] = {0};
    // Find z for all fixed nodes:
    for (shortint i = 0; i < *fixed_queue_size; ++i) {
        shortint current_ind = fixed_queue[*fixed_queue_front];
        *fixed_queue_front = (*fixed_queue_front + 1) % (MAX_N_NODES + 1);
        --(*fixed_queue_size);
        if (current_ind == 0) {
            z[current_ind] = 0;
            v[current_ind] = 0;
        }
        else {
            z[current_ind] = t - value[current_ind];
            v[current_ind] = 1;
        }
        seen[current_ind] = 1;
        *fixed_queue_back = (*fixed_queue_back + 1) % (MAX_N_NODES + 1);
        fixed_queue[*fixed_queue_back] = current_ind;
        ++(*fixed_queue_size);
    }
    
    // Define a branch_stack;
    shortint branch_stack[MAX_N_NODES + 1];
    shortint branch_stack_back = MAX_N_NODES;
    shortint branch_stack_size = 0;
    // Define a fixed_stack;
    shortint fixed_stack[MAX_N_NODES + 1];
    shortint fixed_stack_back = MAX_N_NODES;
    shortint fixed_stack_size = 0;
    // Define a effective_stack;
    shortint effective_stack[MAX_N_NODES + 1];
    shortint effective_stack_back = MAX_N_NODES;
    shortint effective_stack_size = 0;
    // Define a branch_stack_reverse;
    shortint branch_stack_reverse[MAX_N_NODES + 1];
    shortint branch_stack_reverse_back = MAX_N_NODES;
    shortint branch_stack_reverse_size = 0;
    // Define a universal holder:
    shortint holder[MAX_N_NODES + 1];
    shortint holder_front = 0;
    shortint holder_back = MAX_N_NODES;
    shortint holder_size = 0;
    
    // Find z of all branch nodes and stretched free nodes:
    for (shortint i = 0; i < *fixed_queue_size; ++i) {
        shortint current_fixed_ind = fixed_queue[*fixed_queue_front];
        *fixed_queue_front = (*fixed_queue_front + 1) % (MAX_N_NODES + 1);
        --(*fixed_queue_size);
        shortint current_num_kids = num_kids[current_fixed_ind];
        for (shortint j = 0; j < current_num_kids; ++j) {
            shortint current_kid_ind = kids[current_fixed_ind * (MAX_N_NODES + 1) + j];
            if (status[current_kid_ind] != 2) {
                // Reset containers:
                branch_stack_back = MAX_N_NODES;
                branch_stack_size = 0;
                fixed_stack_back = MAX_N_NODES;
                fixed_stack_size = 0;
                effective_stack_back = MAX_N_NODES;
                effective_stack_size = 0;
                branch_stack_reverse_back = MAX_N_NODES;
                branch_stack_reverse_size = 0;
                holder_front = 0;
                holder_back = MAX_N_NODES;
                holder_size = 0;
                // An effective tree for each kids of this fixed node:
                bfs_array_to_fixed(current_kid_ind, status, num_kids, kids, branch_stack, &branch_stack_back, &branch_stack_size, fixed_stack, &fixed_stack_back, &fixed_stack_size);
                // If there is no branch node but there are fixed nodes in this effective tree (which will be a chain):
                if (branch_stack_size == 0 && fixed_stack_size != 0) {
                    // There must be only 1 ending fixed node
                    up_to_node_ind(fixed_stack[fixed_stack_back], current_fixed_ind, status, level, father, num_kids, kids, holder, &holder_front, &holder_back, &holder_size);
                    while (holder_size != 0) {
                        shortint current_node_ind = holder[holder_front];
                        holder_front = (holder_front + 1) % (MAX_N_NODES + 1);
                        --(holder_size);
                        z[current_node_ind] = z[fixed_stack[fixed_stack_back]] + (level[fixed_stack[fixed_stack_back]] - level[current_node_ind]) * (z[current_fixed_ind] - z[fixed_stack[fixed_stack_back]]) / (level[fixed_stack[fixed_stack_back]] - level[current_fixed_ind]);
                        if (current_fixed_ind == 0) {
                            v[current_node_ind] = z[current_node_ind] / z[fixed_stack[fixed_stack_back]];
                        }
                        else {
                            v[current_node_ind] = 1;
                        }
                        seen[current_node_ind] = 1;
                    }
                }
                // If there are branch nodes, do compression:
                else if (branch_stack_size != 0) {
                    // Compressing bottom up in this effective tree:
                    shortint seen_fixed[2*MAX_N_NODES + 1] = {0};
                    shortint eff_ind = dim;
                    while (branch_stack_size != 0) {
                        shortint branch_ind = branch_stack[branch_stack_back];
                        branch_stack_back = (branch_stack_back - 1) % (MAX_N_NODES + 1);
                        --(branch_stack_size);
                        compress_array(branch_ind, value, z, status, level, father, num_kids, kids, seen_fixed, holder, &holder_front, &holder_back, &holder_size, &eff_ind);
                        // Add effective node to kids of its branch node:
                        kids[branch_ind * (MAX_N_NODES + 1) + num_kids[branch_ind]] = eff_ind;
                        effective_stack_back = (effective_stack_back + 1) % (MAX_N_NODES + 1);
                        effective_stack[effective_stack_back] = eff_ind;
                        ++(effective_stack_size);
                        branch_stack_reverse_back = (branch_stack_reverse_back + 1) % (MAX_N_NODES + 1);
                        branch_stack_reverse[branch_stack_reverse_back] = branch_ind;
                        ++(branch_stack_reverse_size);
                    }
                    // Note: bijection between branch_stack and effective_stack
                    // Find z of the top most branch node:
                    shortint effective_ind = effective_stack[effective_stack_back];
                    effective_stack_back = (effective_stack_back - 1) % (MAX_N_NODES + 1);
                    --(effective_stack_size);
                    shortint branch_ind = branch_stack_reverse[branch_stack_reverse_back];
                    branch_stack_reverse_back = (branch_stack_reverse_back - 1) % (MAX_N_NODES + 1);
                    --(branch_stack_reverse_size);
                    realnumber dist = (level[branch_ind] - level[current_fixed_ind]) + 1.0 / value[effective_ind];
                    realnumber unit_len = (z[current_fixed_ind] - z[effective_ind]) / dist;
                    z[branch_ind] = z[effective_ind] + (1.0 / value[effective_ind]) * unit_len;
                    if (current_fixed_ind == 0) {
                        v[branch_ind] = z[branch_ind] / z[effective_ind];
                    }
                    else {
                        v[branch_ind] = 1;
                    }
                    // ! Remove effective node from tree !:
                    kids[branch_ind * (MAX_N_NODES + 1) + num_kids[branch_ind]] = -2;
                    seen[branch_ind] = 1;
                    // Find z of free nodes between top most branch node and root of current effective tree:
                    up_to_node_ind(branch_ind, current_fixed_ind, status, level, father, num_kids, kids, holder, &holder_front, &holder_back, &holder_size);
                    while (holder_size != 0) {
                        shortint current_node_ind = holder[holder_front];
                        holder_front = (holder_front + 1) % (MAX_N_NODES + 1);
                        --(holder_size);
                        z[current_node_ind] = z[branch_ind] + (level[branch_ind] - level[current_node_ind]) * unit_len;
                        if (current_fixed_ind == 0) {
                            v[current_node_ind] = z[current_node_ind] / z[branch_ind];
                        }
                        else {
                            v[current_node_ind] = 1;
                        }
                        seen[current_node_ind] = 1;
                    }
                    // Decompressing:
                    while (effective_stack_size != 0) {
                        shortint current_eff_ind = effective_stack[effective_stack_back];
                        effective_stack_back = (effective_stack_back - 1) % (MAX_N_NODES + 1);
                        --(effective_stack_size);
                        shortint current_branch_ind = branch_stack_reverse[branch_stack_reverse_back];
                        branch_stack_reverse_back = (branch_stack_reverse_back - 1) % (MAX_N_NODES + 1);
                        --(branch_stack_reverse_size);
                        up_to_branch_ind(current_branch_ind, status, level, father, num_kids, kids, holder, &holder_front, &holder_back, &holder_size);
                        dist = (level[current_branch_ind] - level[holder[holder_back]]) + 1.0 / value[current_eff_ind];
                        unit_len = (z[holder[holder_back]] - z[current_eff_ind]) / dist;
                        z[current_branch_ind] = z[current_eff_ind] + (1.0 / value[current_eff_ind]) * unit_len;
                        if (current_fixed_ind == 0) {
                            v[current_branch_ind] = z[current_branch_ind] / z[current_eff_ind];
                        }
                        else {
                            v[current_branch_ind] = 1;
                        }
                        seen[current_branch_ind] = 1;
                        // Find z of all the free nodes between current branch node and its closest up branch node:
                        holder_front = (holder_front + 1) % (MAX_N_NODES + 1);
                        --(holder_size);
                        while (holder_size != 0) {
                            shortint free_ind = holder[holder_front];
                            holder_front = (holder_front + 1) % (MAX_N_NODES + 1);
                            --(holder_size);
                            z[free_ind] = z[current_branch_ind] + (level[current_branch_ind] - level[free_ind]) * unit_len;
                            if (current_fixed_ind == 0) {
                                v[free_ind] = z[free_ind] / z[current_branch_ind];
                            }
                            else {
                                v[free_ind] = 1;
                            }
                            seen[free_ind] = 1;
                        }
                        // ! Remove effective node from tree !:
                        kids[current_branch_ind * (MAX_N_NODES + 1) + num_kids[current_branch_ind]] = -2;
                    }
                    // Find z of the rest stretched free nodes in current effective tree:
                    while (fixed_stack_size != 0) {
                        shortint a_fixed_ind = fixed_stack[fixed_stack_back];
                        fixed_stack_back = (fixed_stack_back - 1) % (MAX_N_NODES + 1);
                        --(fixed_stack_size);
                        if (a_fixed_ind != 0) {
                            up_to_branch_ind(a_fixed_ind, status, level, father, num_kids, kids, holder, &holder_front, &holder_back, &holder_size);
                            shortint holder_back_ind = holder[holder_back];
                            while (holder_size != 0) {
                                shortint a_free_ind = holder[holder_front];
                                holder_front = (holder_front + 1) % (MAX_N_NODES + 1);
                                --(holder_size);
                                z[a_free_ind] = z[a_fixed_ind] + (level[a_fixed_ind] - level[a_free_ind]) * (z[holder_back_ind] - z[a_fixed_ind]) / (level[a_fixed_ind] - level[holder_back_ind]);
                                if (current_fixed_ind == 0) {
                                    v[a_free_ind] = z[a_free_ind] / z[a_fixed_ind];
                                }
                                else {
                                    v[a_free_ind] = 1;
                                }
                                seen[a_free_ind] = 1;
                            }
                        }
                    }
                }
            }
        }
        // Add the current fixed node back to keep fixed_queue intact:
        *fixed_queue_back = (*fixed_queue_back + 1) % (MAX_N_NODES + 1);
        fixed_queue[*fixed_queue_back] = current_fixed_ind;
        ++(*fixed_queue_size);
    }
    // Now we have seen all those stretched nodes
    // Find z of all other (free) nodes (unstretched):
    bfs_array_unstretched(status, level, father, num_kids, kids, seen, z, v);
    v[0] = 0;
}

/*
 This function computes the next turing point, then the next node to be fixed and store its index in t_data[1]:
 */
__host__ __device__ void next_turn_array(realnumber t_data[], realnumber t_pre, realnumber z_pre[], realnumber v[], realnumber value[], shortint status[], shortint level[], shortint dim) {
    v[0] = 0;
    realnumber t_possilbe[MAX_N_NODES + 1];
    for (shortint i = 0; i <= dim; ++i) {
        t_possilbe[i] = -INFINITY;
    }
    // Find intersections:
    for (shortint i = 1; i <= dim; ++i) {
        if (status[i] != 2) {
            t_possilbe[i] = (z_pre[i] - v[i]*t_pre + value[i]) / (1 - v[i]);
        }
    }
    // Find the max t among all intersections:
    realnumber max = t_possilbe[0];
    shortint max_ind = 0;
    for (shortint i = 1; i <= dim; ++i) {
        if (t_possilbe[i] >= max) {
            max = t_possilbe[i];
        }
    }
    shortint top_level = dim + 1;
    for (shortint i = 1; i <= dim; ++i) {
        if (t_possilbe[i] == max && level[i] < top_level) {
            max_ind = i;
            top_level = level[i]; // Fix the node with smallest level first
        }
    }
    realnumber min_v = 1;
    for (shortint i = 1; i <= dim; ++i) {
        if (v[i] <= min_v) {
            min_v = v[i];
        }
    }
    // If there won't be any new fixed node:
    if (min_v == 1) {
        t_data[0] = -INFINITY;
        t_data[1] = 0;
    }
    // If there will be a next fixed node:
    else {
        t_data[0] = max;
        t_data[1] = max_ind;
    }
}

/*
 This function computes the value of df(t) at t:
 */
__host__ __device__ realnumber df_exact_array(shortint father[], shortint num_kids[], shortint kids[], realnumber z_pre[], realnumber v[], shortint dim) {
    v[0] = 0;
    realnumber df = 0;
    for (shortint i = 1; i <= dim; ++i) {
        realnumber holder = z_pre[i] - z_pre[father[i]];
        for (shortint j = 0; j < num_kids[i]; ++j) {
            holder -= z_pre[kids[i * (MAX_N_NODES + 1) + j]] - z_pre[i];
        }
        df += holder * v[i];
    }
    return df;
}

/*
 This function uses all the above functions to look for solutions to df(t) = -1:
 */
__host__ __device__ void best_tree_array(realnumber value[], realnumber z[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim) {
    shortint level[MAX_N_NODES + 1] = {0};
    realnumber v[MAX_N_NODES + 1] = {1};
    realnumber z_pre[2*MAX_N_NODES + 1]; // For computing v[]
    for (shortint i = 0; i <= dim; ++i) {
        z[i] = 0;
        level[i] = 0;
        v[i] = 1;
        z_pre[i] = z[i];
    }
    v[0] = 0;
    shortint fixed_queue[MAX_N_NODES + 1];
    shortint fixed_queue_front = 0;
    shortint fixed_queue_back = MAX_N_NODES;
    shortint fixed_queue_size = 0;
    bfs_get_level(status, level, num_kids, kids, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size);
    // First turning point:
    realnumber n_max = value[1];
    shortint ind = 0;
    for (shortint i = 1; i <= dim; ++i) {
        if (value[i] >= n_max) {
            n_max = value[i];
        }
    }
    shortint top_level = dim + 1;
    for (shortint i = 1; i <= dim; ++i) {
        if (value[i] == n_max && level[i] < top_level) {
            ind = i;
            top_level = level[i]; // Fix the node with smallest level first
        }
    }
    status[ind] = 2;
    fixed_queue_back = (fixed_queue_back + 1) % (MAX_N_NODES + 1);
    fixed_queue[fixed_queue_back] = ind;
    ++(fixed_queue_size);
    realnumber t_now = n_max;
    realnumber t_pre = n_max;
    // Update speed by looking ahead wiht CURRENT status:
    update_z_array(t_now-delta, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
    realnumber df_now = 0;
    realnumber df_pre = 0;
    // Other turning points:
    shortint next_possible_turn_current_status = 0;
    while (df_now > -1 && t_now > -INFINITY) {
        t_pre = t_now;
        df_pre = df_now;
        realnumber t_data[2] = {0};
        next_turn_array(t_data, t_pre, z_pre, v, value, status, level, dim);
        t_now = t_data[0];
        if (t_now == -INFINITY) {
            break;
        }
        ind = t_data[1];
        next_possible_turn_current_status = status[ind];
        status[ind] = 2;
        fixed_queue_back = (fixed_queue_back + 1) % (MAX_N_NODES + 1);
        fixed_queue[fixed_queue_back] = ind;
        ++(fixed_queue_size);
        // Update branching status of nodes:
        for (shortint i = 1; i <= dim; ++i) {
            if (is_branch_array(i, status, father, num_kids, kids, dim) == 1) {
                status[i] = 1;
            }
        }
        // Update z:
        update_z_array(t_now, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        for (shortint i = 1; i <= dim; ++i) {
            z_pre[i] = z[i];
        }
        df_now = df_exact_array(father, num_kids, kids, z_pre, v, dim);
        // Update speed by looking ahead wiht CURRENT status:
        update_z_array(t_now-delta, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        /*
         for (shortint i = 0; i <= dim; ++i) {
         printf("(%d, %f, %f)\n", i, z_pre[i], v[i]);
         }
         printf("________\n");
         */
    }
    // Finally get what we really want:
    if (t_now != -INFINITY) {
        // Reverse the last fixed node to its previous status:
        status[ind] = next_possible_turn_current_status;
        fixed_queue_back = (fixed_queue_back - 1) % (MAX_N_NODES + 1);
        --(fixed_queue_size);
        realnumber k = (df_pre - df_now) / (t_pre - t_now);
        realnumber t = t_pre - (df_pre + 1)  / k;
        update_z_array(t, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        z[0] = t; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
    }
    else {
        realnumber t_last = t_pre - delta;
        update_z_array(t_last, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        for (shortint i = 1; i <= dim; ++i) {
            z_pre[i] = z[i];
        }
        realnumber df_last = df_exact_array(father, num_kids, kids, z_pre, v, dim);
        realnumber k = (df_pre - df_last) / (t_pre - t_last);
        realnumber t = (k*t_pre - df_pre - 1) / k;
        update_z_array(t, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        z[0] = t; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
    }
}

/*
 This function uses all the above functions to look for solutions to df(t) = -1, but with slightly different constraints:
 */
__host__ __device__ void best_tree_array_inner(realnumber value[], realnumber z[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim) {
    shortint level[MAX_N_NODES + 1] = {0};
    realnumber v[MAX_N_NODES + 1] = {1};
    realnumber z_pre[2*MAX_N_NODES + 1]; // For computing v[]
    for (shortint i = 0; i <= dim; ++i) {
        z[i] = 0;
        level[i] = 0;
        v[i] = 1;
        z_pre[i] = z[i];
    }
    v[0] = 0;
    shortint fixed_queue[MAX_N_NODES + 1];
    shortint fixed_queue_front = 0;
    shortint fixed_queue_back = MAX_N_NODES;
    shortint fixed_queue_size = 0;
    bfs_get_level(status, level, num_kids, kids, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size);
    // First turning point:
    realnumber n_max = value[1];
    shortint ind = 0;
    for (shortint i = 1; i <= dim; ++i) {
        if (value[i] >= n_max) {
            n_max = value[i];
        }
    }
    shortint top_level = dim + 1;
    for (shortint i = 1; i <= dim; ++i) {
        if (value[i] == n_max && level[i] < top_level) {
            ind = i;
            top_level = level[i]; // Fix the node with smallest level first
        }
    }
    status[ind] = 2;
    fixed_queue_back = (fixed_queue_back + 1) % (MAX_N_NODES + 1);
    fixed_queue[fixed_queue_back] = ind;
    ++(fixed_queue_size);
    realnumber t_now = n_max;
    realnumber t_pre = n_max;
    // Update speed by looking ahead wiht CURRENT status:
    update_z_array(t_now-delta, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
    realnumber df_now = 0;
    realnumber df_pre = 0;
    // Other turning points:
    shortint next_possible_turn_current_status = 0;
    while (df_now > -1 && t_now > -INFINITY) {
        t_pre = t_now;
        df_pre = df_now;
        realnumber t_data[2] = {0};
        next_turn_array(t_data, t_pre, z_pre, v, value, status, level, dim);
        t_now = t_data[0];
        if (t_now == -INFINITY) {
            break;
        }
        ind = t_data[1];
        next_possible_turn_current_status = status[ind];
        status[ind] = 2;
        fixed_queue_back = (fixed_queue_back + 1) % (MAX_N_NODES + 1);
        fixed_queue[fixed_queue_back] = ind;
        ++(fixed_queue_size);
        // Update branching status of nodes:
        for (shortint i = 1; i <= dim; ++i) {
            if (is_branch_array(i, status, father, num_kids, kids, dim) == 1) {
                status[i] = 1;
            }
        }
        // Update z:
        update_z_array(t_now, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        for (shortint i = 1; i <= dim; ++i) {
            z_pre[i] = z[i];
        }
        df_now = df_exact_array(father, num_kids, kids, z_pre, v, dim);
        if (t_now < 0) {
            break;
        }
        // Update speed by looking ahead wiht CURRENT status:
        update_z_array(t_now-delta, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
    }
    // Finally get what we really want:
    if (df_now > -1 && t_now != -INFINITY) {
        // Reverse the last fixed node to its previous status:
        status[ind] = next_possible_turn_current_status;
        fixed_queue_back = (fixed_queue_back - 1) % (MAX_N_NODES + 1);
        --(fixed_queue_size);
        update_z_array(0, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        z[0] = 0; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
    }
    else if (t_now != -INFINITY) {
        // Reverse the last fixed node to its previous status:
        status[ind] = next_possible_turn_current_status;
        fixed_queue_back = (fixed_queue_back - 1) % (MAX_N_NODES + 1);
        --(fixed_queue_size);
        realnumber k = (df_pre - df_now) / (t_pre - t_now);
        realnumber t = t_pre - (df_pre + 1)  / k;
        if (t > 0) {
            update_z_array(t, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
            z[0] = t; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
        }
        else {
            update_z_array(0, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
            z[0] = 0; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
        }
    }
    else {
        realnumber t_last = t_pre - delta;
        update_z_array(t_last, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
        for (shortint i = 1; i <= dim; ++i) {
            z_pre[i] = z[i];
        }
        realnumber df_last = df_exact_array(father, num_kids, kids, z_pre, v, dim);
        realnumber k = (df_pre - df_last) / (t_pre - t_last);
        realnumber t = (k*t_pre - df_pre - 1) / k;
        if (t > 0) {
            update_z_array(t, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
            z[0] = t; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
        }
        else {
            update_z_array(0, v, fixed_queue, &fixed_queue_front, &fixed_queue_back, &fixed_queue_size, value, z, status, level, father, num_kids, kids, dim);
            z[0] = 0; // Covention: z[0] = t, and z[i] = tree[i]->z for i != 0
        }
    }
}

//
//
//
//
//
//****************************** END OF BEI's CODE WITH ARRAYS ******************************
//
//
//






//
//
//
//****************************** START OF BEI'S NEW CODE ******************************
//
//
//
//
//
// !New! status: 0 = free, 1 = unstretched, 2 = fixed

__host__ __device__ void heap_delete(shortint node_index, realnumber max_heap[], shortint heap_to_tree_index[], shortint tree_to_heap_index[], shortint *heap_size) {
    //clock_t begin = clock();
    // Remove the original value on heap corresponding to node_index:
    shortint current_ind = tree_to_heap_index[node_index];
    realnumber temp = max_heap[*heap_size - 1];
    max_heap[current_ind] = temp;
    shortint temp_node_ind = heap_to_tree_index[*heap_size - 1];
    heap_to_tree_index[current_ind] = temp_node_ind;
    tree_to_heap_index[temp_node_ind] = current_ind;
    // current_ind <---> temp_node_ind
    // Reduce heap size now:
    --(*heap_size);
    if (max_heap[current_ind] > max_heap[(current_ind - 1) / 2]) {
        // Bubble up:
        while (1) {
            shortint father_ind = (current_ind - 1) / 2; // Integer division acts like floor function
            if (max_heap[current_ind] > max_heap[father_ind]) {
                max_heap[current_ind] = max_heap[father_ind];
                max_heap[father_ind] = temp;
                heap_to_tree_index[current_ind] = heap_to_tree_index[father_ind];
                heap_to_tree_index[father_ind] = temp_node_ind;
                tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                tree_to_heap_index[temp_node_ind] = father_ind;
                current_ind = father_ind;
            }
            else {
                break;
            }
        }
    }
    else {
        // Bubble down:
        while (1) {
            shortint left_ind = 2 * current_ind + 1;
            shortint right_ind = 2 * current_ind + 2;
            // Make sure we are reaching within boundary:
            if (left_ind < *heap_size && right_ind < *heap_size) {
                if (temp < max_heap[left_ind] && temp < max_heap[right_ind]) {
                    if (max_heap[left_ind] >= max_heap[right_ind]) {
                        max_heap[current_ind] = max_heap[left_ind];
                        max_heap[left_ind] = temp;
                        heap_to_tree_index[current_ind] = heap_to_tree_index[left_ind];
                        heap_to_tree_index[left_ind] = temp_node_ind;
                        tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                        tree_to_heap_index[temp_node_ind] = left_ind;
                        current_ind = left_ind;
                    }
                    else {
                        max_heap[current_ind] = max_heap[right_ind];
                        max_heap[right_ind] = temp;
                        heap_to_tree_index[current_ind] = heap_to_tree_index[right_ind];
                        heap_to_tree_index[right_ind] = temp_node_ind;
                        tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                        tree_to_heap_index[temp_node_ind] = right_ind;
                        current_ind = right_ind;
                    }
                }
                else if (temp < max_heap[left_ind] && temp >= max_heap[right_ind]) {
                    max_heap[current_ind] = max_heap[left_ind];
                    max_heap[left_ind] = temp;
                    heap_to_tree_index[current_ind] = heap_to_tree_index[left_ind];
                    heap_to_tree_index[left_ind] = temp_node_ind;
                    tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                    tree_to_heap_index[temp_node_ind] = left_ind;
                    current_ind = left_ind;
                }
                else if (temp < max_heap[right_ind] && temp >= max_heap[left_ind]) {
                    max_heap[current_ind] = max_heap[right_ind];
                    max_heap[right_ind] = temp;
                    heap_to_tree_index[current_ind] = heap_to_tree_index[right_ind];
                    heap_to_tree_index[right_ind] = temp_node_ind;
                    tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                    tree_to_heap_index[temp_node_ind] = right_ind;
                    current_ind = right_ind;
                }
                else {
                    break;
                }
            }
            else if (left_ind < *heap_size && temp < max_heap[left_ind]) {
                max_heap[current_ind] = max_heap[left_ind];
                max_heap[left_ind] = temp;
                heap_to_tree_index[current_ind] = heap_to_tree_index[left_ind];
                heap_to_tree_index[left_ind] = temp_node_ind;
                tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                tree_to_heap_index[temp_node_ind] = left_ind;
                current_ind = left_ind;
            }
            else if (right_ind < *heap_size && temp < max_heap[right_ind]) {
                max_heap[current_ind] = max_heap[right_ind];
                max_heap[right_ind] = temp;
                heap_to_tree_index[current_ind] = heap_to_tree_index[right_ind];
                heap_to_tree_index[right_ind] = temp_node_ind;
                tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
                tree_to_heap_index[temp_node_ind] = right_ind;
                current_ind = right_ind;
            }
            else {
                break;
            }
        }
    }
    //clock_t end = clock();
    //realnumber cpu_time = (realnumber)(end - begin) / CLOCKS_PER_SEC;
    //heap_time2 += cpu_time;
}

__host__ __device__ void heap_insert(realnumber intersetion, shortint node_index, realnumber max_heap[], shortint heap_to_tree_index[], shortint tree_to_heap_index[], shortint *heap_size) {
    //clock_t begin = clock();
    // Increase size first:
    ++(*heap_size);
    max_heap[*heap_size - 1] = intersetion;
    heap_to_tree_index[*heap_size - 1] = node_index;
    tree_to_heap_index[node_index] = *heap_size - 1;
    shortint current_ind = *heap_size - 1;
    // Bubble up:
    while (1) {
        shortint father_ind = (current_ind - 1) / 2; // Integer division acts like floor function
        if (intersetion > max_heap[father_ind]) {
            max_heap[current_ind] = max_heap[father_ind];
            max_heap[father_ind] = intersetion;
            heap_to_tree_index[current_ind] = heap_to_tree_index[father_ind];
            heap_to_tree_index[father_ind] = node_index;
            tree_to_heap_index[heap_to_tree_index[current_ind]] = current_ind;
            tree_to_heap_index[node_index] = father_ind;
            current_ind = father_ind;
        }
        else {
            break;
        }
    }
    //clock_t end = clock();
    //realnumber cpu_time = (realnumber)(end - begin) / CLOCKS_PER_SEC;
    //heap_time2 += cpu_time;
}

// Find the path to the closest fixed node above:
__host__ __device__ void up_to_fixed_ind(shortint start_ind, shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    shortint current_ind = father[start_ind];
    if (current_ind < 0) {
        return;
    }
    *holder_front = 0;
    *holder_back = 0;
    *holder_size = 1;
    holder[*holder_back] = current_ind;
    while (status[current_ind] != 2) {
        current_ind = father[current_ind];
        if (current_ind < 0) {
            return;
        }
        *holder_back = (*holder_back + 1) % (MAX_N_NODES + 1);
        holder[*holder_back] = current_ind;
        ++(*holder_size);
        //printf("%d ", holder[*holder_back]);
    }
}

// BFS on entire tree to get levels:
__host__ __device__ void bfs_get_level_new(shortint status[], shortint level[], shortint num_kids[], shortint kids[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size) {
    // Reset containter:
    *queue_front = 0;
    queue[*queue_front] = 0;
    *queue_back = 0;
    *queue_size = 1;
    level[0] = 0;
    while (*queue_size != 0) {
        shortint current_ind = queue[*queue_front];
        *queue_front = (*queue_front + 1) % (MAX_N_NODES + 1);
        --*queue_size;
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            // Note: this BFS will never look at effective nodes produced during compression
            *queue_back = (*queue_back + 1) % (MAX_N_NODES + 1);
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            queue[*queue_back] = kid_ind;
            ++*queue_size;
            level[kid_ind] = level[current_ind] + 1;
        }
    }
}

// BFS up to fixed nodes and get all nodes and store in nodes_stack:
__host__ __device__ void bfs_to_fixed_get_all_new(shortint start_ind, shortint status[], shortint num_kids[], shortint kids[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint node_stack[], shortint *node_stack_back, shortint *node_stack_size) {
    // Reset container:
    *queue_front = 0;
    queue[*queue_front] = start_ind;
    *queue_back = 0;
    *queue_size = 1;
    while (*queue_size != 0) {
        shortint current_ind = queue[*queue_front];
        *queue_front = (*queue_front + 1) % (MAX_N_NODES + 1);
        --*queue_size;
        *node_stack_back = (*node_stack_back + 1) % (MAX_N_NODES + 1);
        node_stack[*node_stack_back] = current_ind;
        ++(*node_stack_size);
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            // Note: this BFS will never look at effective nodes produced during compression
            // We stop when we see fixed or effective nodes
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            if (status[kid_ind] != 2 && status[kid_ind] != 3) {
                *queue_back = (*queue_back + 1) % (MAX_N_NODES + 1);
                queue[*queue_back] = kid_ind;
                ++*queue_size;
            }
        }
    }
}

__host__ __device__ void update_z_new(shortint new_fixed_ind, realnumber value[], realnumber t, realnumber z[], realnumber z_timestamp[], realnumber v[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    // Reset holder:
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    // Subtree above new_fixed_ind:
    if (father[new_fixed_ind] != 0) {
        up_to_fixed_ind(new_fixed_ind, status, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
        shortint current_root_ind = holder[(*holder_back - 1) % (MAX_N_NODES + 1)]; // The second last element of holder is the root of the current subtree
        // Reset holder:
        *holder_front = 0;
        *holder_back = (MAX_N_NODES + 1) - 1;
        *holder_size = 0;
        bfs_to_fixed_get_all_new(current_root_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
    }
    // Subtrees below new_fixed_ind:
    shortint current_num_kids = num_kids[new_fixed_ind];
    for (shortint i = 0; i < current_num_kids; ++i) {
        shortint current_root_ind = kids[new_fixed_ind * (MAX_N_NODES + 1) + i];
        bfs_to_fixed_get_all_new(current_root_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
    }
    // Update z of nodes in relevant subtrees before updating v:
    while (*holder_size != 0) {
        shortint ind = holder[*holder_back];
        *holder_back = (*holder_back - 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        z[ind] = z[ind] + v[ind] * (t - z_timestamp[ind]);
        z_timestamp[ind] = t;
    }
    z[new_fixed_ind] = z[new_fixed_ind] + v[new_fixed_ind] * (t - z_timestamp[new_fixed_ind]);
    z_timestamp[new_fixed_ind] = t;
}

__host__ __device__ void update_v_old_new(shortint new_fixed_ind, shortint pre_fixed_ind, realnumber t, realnumber v[], realnumber v_old[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim, shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    // Reset holder:
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    // Find all nodes in the subtree above new_fixed_ind:
    if (father[new_fixed_ind] != 0) {
        up_to_fixed_ind(new_fixed_ind, status, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
        shortint current_root_ind = holder[(*holder_back - 1) % (MAX_N_NODES + 1)];
        // Reset holder:
        *holder_front = 0;
        *holder_back = (MAX_N_NODES + 1) - 1;
        *holder_size = 0;
        // Find all nodes in above subtree:
        bfs_to_fixed_get_all_new(current_root_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
    }
    // Find all nodes in below subtrees:
    shortint current_num_kids = num_kids[new_fixed_ind];
    for (shortint i = 0; i < current_num_kids; ++i) {
        shortint current_kid_ind = kids[new_fixed_ind * (MAX_N_NODES + 1) + i];
        bfs_to_fixed_get_all_new(current_kid_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
    }
    // Now holder contains all other nodes needed
    while (*holder_size != 0) {
        shortint current_node_ind = holder[*holder_front];
        *holder_front = (*holder_front + 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        v_old[current_node_ind] = v[current_node_ind];
    }
    v_old[new_fixed_ind] = v[new_fixed_ind];
    v_old[pre_fixed_ind] = v[pre_fixed_ind];
}

// Idea: we label each subtree by its root's index, so we can heapify subtrees only
__host__ __device__ shortint next_turn_better_new(shortint new_fixed_ind, realnumber t_pre, realnumber z[], realnumber v[], realnumber value[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim, shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size, realnumber max_heap[], shortint heap_to_tree_index[], shortint tree_to_heap_index[], shortint *heap_size) {
    v[0] = 0;
    // Delete subtree with root new_fixed_node if it is in the heap:
    if (tree_to_heap_index[new_fixed_ind] < *heap_size) {
        heap_delete(new_fixed_ind, max_heap, heap_to_tree_index, tree_to_heap_index, heap_size);
    }
    // Reset holder:
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    // Find all nodes in the subtree above new_fixed_ind:
    if (father[new_fixed_ind] != 0) {
        up_to_fixed_ind(new_fixed_ind, status, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
        shortint current_root_ind = holder[(*holder_back - 1) % (MAX_N_NODES + 1)];
        // Delete the previsoue subtree with current_root_ind as root:
        heap_delete(current_root_ind, max_heap, heap_to_tree_index, tree_to_heap_index, heap_size);
        // Reset holder:
        *holder_front = 0;
        *holder_back = (MAX_N_NODES + 1) - 1;
        *holder_size = 0;
        // Find all nodes in above subtree:
        bfs_to_fixed_get_all_new(current_root_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
        realnumber t_possilbe = -INFINITY;
        while (*holder_size != 0) {
            shortint current_node_ind = holder[*holder_front];
            *holder_front = (*holder_front + 1) % (MAX_N_NODES + 1);
            --(*holder_size);
            // Find max intersection:
            if (status[current_node_ind] != 2) {
                realnumber curr_t_possilbe = (z[current_node_ind] - v[current_node_ind] * t_pre + value[current_node_ind]) / (1 - v[current_node_ind]);
                if (t_possilbe < curr_t_possilbe) {
                    t_possilbe = curr_t_possilbe;
                }
            }
        }
        // Insert max intersection into heap:
        heap_insert(t_possilbe, current_root_ind, max_heap, heap_to_tree_index, tree_to_heap_index, heap_size);
    }
    // Find all nodes in below subtrees:
    int current_num_kids = num_kids[new_fixed_ind];
    for (shortint i = 0; i < current_num_kids; ++i) {
        // Reset holder:
        *holder_front = 0;
        *holder_back = (MAX_N_NODES + 1) - 1;
        *holder_size = 0;
        shortint current_kid_ind = kids[new_fixed_ind * (MAX_N_NODES + 1) + i];
        realnumber t_possilbe = -INFINITY;
        if (status[current_kid_ind] != 2) {
            bfs_to_fixed_get_all_new(current_kid_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
            while (*holder_size != 0) {
                shortint current_node_ind = holder[*holder_front];
                *holder_front = (*holder_front + 1) % (MAX_N_NODES + 1);
                --(*holder_size);
                // Find max intersection:
                if (status[current_node_ind] != 2) {
                    realnumber curr_t_possilbe = (z[current_node_ind] - v[current_node_ind] * t_pre + value[current_node_ind]) / (1 - v[current_node_ind]);
                    if (t_possilbe < curr_t_possilbe) {
                        t_possilbe = curr_t_possilbe;
                    }
                }
            }
        }
        // Insert into heap:
        heap_insert(t_possilbe, current_kid_ind, max_heap, heap_to_tree_index, tree_to_heap_index, heap_size);
    }
    // Find the next fixed node:
    shortint max_subtree_root = heap_to_tree_index[0];
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    bfs_to_fixed_get_all_new(max_subtree_root, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
    realnumber t_want = max_heap[0];
    while (*holder_size != 0) {
        shortint current_node_ind = holder[*holder_front];
        *holder_front = (*holder_front + 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        // Find max intersection:
        if (status[current_node_ind] != 2) {
            realnumber curr_t_possilbe = (z[current_node_ind] - v[current_node_ind] * t_pre + value[current_node_ind]) / (1 - v[current_node_ind]);
            if (t_want == curr_t_possilbe) {
                new_fixed_ind = current_node_ind;
            }
        }
    }
    return new_fixed_ind;
}

// "Prune" (using BFS) unstretched nodes of a given subtree by setting their status be to 1:
__host__ __device__ void prune(shortint root, shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    // Reset containters:
    *queue_front = 0;
    queue[*queue_front] = root;
    *queue_back = 0;
    *queue_size = 1;
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    shortint leaf_count = 0;
    // BFS to find all nodes:
    while (*queue_size != 0) {
        shortint current_ind = queue[*queue_front];
        *queue_front = (*queue_front + 1) % (MAX_N_NODES + 1);
        --*queue_size;
        *holder_back = (*holder_back + 1) % (MAX_N_NODES + 1);
        holder[*holder_back] = current_ind;
        ++(*holder_size);
        if (num_kids[current_ind] == 0) {
            ++leaf_count;
        }
        else if (num_kids[current_ind] == 1 && status[current_ind] == 2) {
            ++leaf_count;
        }
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            if (status[kid_ind] != 2) {
                *queue_back = (*queue_back + 1) % (MAX_N_NODES + 1);
                queue[*queue_back] = kid_ind;
                ++*queue_size;
            }
        }
    }
    // Now holder contains all nodes on the current subtree
    while (*holder_size != 0) {
        // It is important to start from the back of holder!
        shortint current_ind = holder[*holder_back];
        *holder_back = (*holder_back - 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        // Only prune unfixed nodes:
        if (status[current_ind] != 2) {
            // Set true leaves to be unstretched:
            if (leaf_count != 0) {
                --leaf_count;
                if (num_kids[current_ind] == 0) {
                    status[current_ind] = 1;
                }
            }
            // Change the status of other unstreched nodes:
            else {
                shortint unstretched_indicator = 1;
                for (shortint i = 0; i < num_kids[current_ind]; ++i) {
                    shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
                    if (status[kid_ind] != 1) {
                        unstretched_indicator = 0;
                        break;
                    }
                }
                if (unstretched_indicator == 1) {
                    status[current_ind] = 1;
                }
            }
        }
    }
}

// Update (using BFS + DFS) stretched nodes of a given subtree:
__host__ __device__ void update_v_reborn_stretched(shortint root, realnumber v[], realnumber gamma[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint seen[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size) {
    // Reset containers:
    *queue_front = 0;
    queue[*queue_front] = root;
    *queue_back = 0;
    *queue_size = 1;
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    // Use BFS to put all stretched free nodes into holder:
    while (*queue_size != 0) {
        shortint current_ind = queue[*queue_front];
        *queue_front = (*queue_front + 1) % (MAX_N_NODES + 1);
        --(*queue_size);
        // Don't look at unstretched nodes nor fixed nodes:
        if (status[current_ind] == 0) {
            *holder_back = (*holder_back + 1) % (MAX_N_NODES + 1);
            holder[*holder_back] = current_ind;
            ++(*holder_size);
        }
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            if (status[kid_ind] != 2) {
                *queue_back = (*queue_back + 1) % (MAX_N_NODES + 1);
                queue[*queue_back] = kid_ind;
                ++*queue_size;
            }
        }
    }
    //printf("Unstretched nodes: ");
    //for (int i = 0; i < *holder_size; ++i) {
    //printf("%d ", holder[i]);
    //}
    //printf("\n");
    // Now holder contains all stretched free nodes
    // Compression from bottom-right:
    while (*holder_size != 1) {
        int current_ind = holder[*holder_back];
        *holder_back = (*holder_back - 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        seen[current_ind] = 1;
        realnumber numerator = 0, denominator = 0;
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            // Don't look at unstretched nodes:
            if (status[kid_ind] != 1) {
                numerator += gamma[kid_ind] * v[kid_ind];
                denominator += gamma[kid_ind];
            }
        }
        v[current_ind] = numerator / denominator;
        gamma[current_ind] = 1 / ((1 / gamma[current_ind]) + (1 / denominator));
    }
    // Reset holder (now acting as a stack for DFS):
    *holder_front = 0;
    *holder_back = 0;
    holder[*holder_back] = root;
    *holder_size = 1;
    // Decompression using DFS from the top:
    while (*holder_size != 0) {
        shortint current_ind = holder[*holder_back];
        *holder_back = (*holder_back - 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        //printf("current_ind = %d\n", current_ind);
        shortint parent_ind = father[current_ind];
        gamma[current_ind] = 1; // !MUST RESET gamma here!
        realnumber numerator = gamma[current_ind] * v[parent_ind], denominator = gamma[current_ind];
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            // Don't look at unstretched nodes:
            if (status[kid_ind] != 1) {
                numerator += gamma[kid_ind] * v[kid_ind];
                denominator += gamma[kid_ind];
            }
        }
        v[current_ind] = numerator / denominator;
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            // Don't look at unstretched nodes nor fixed nodes:
            if (status[kid_ind] == 0) {
                *holder_back = (*holder_back + 1) % (MAX_N_NODES + 1);
                holder[*holder_back] = kid_ind;
                ++(*holder_size);
            }
        }
    }
}

// Update unstretched nodes of a given subtree:
__host__ __device__ void update_v_reborn_unstretched(shortint root, realnumber v[], realnumber gamma[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size) {
    // Reset containters:
    *queue_front = 0;
    queue[*queue_front] = root;
    *queue_back = 0;
    *queue_size = 1;
    // Use BFS to set v for all unstretched nodes:
    while (*queue_size != 0) {
        shortint current_ind = queue[*queue_front];
        *queue_front = (*queue_front + 1) % (MAX_N_NODES + 1);
        --*queue_size;
        if (status[current_ind] == 1) {
            v[current_ind] = v[father[current_ind]];
        }
        for (shortint i = 0; i < num_kids[current_ind]; ++i) {
            shortint kid_ind = kids[current_ind * (MAX_N_NODES + 1) + i];
            if (status[kid_ind] != 2) {
                *queue_back = (*queue_back + 1) % (MAX_N_NODES + 1);
                queue[*queue_back] = kid_ind;
                ++*queue_size;
            }
        }
    }
}

// Update v for all nodes that are affected at each round:
__host__ __device__ void update_v_reborn(shortint new_fixed_ind, realnumber v[], realnumber gamma[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size, shortint seen[]) {
    v[0] = 0;
    v[new_fixed_ind] = 1;
    // Reset containers:
    *queue_front = 0;
    *queue_back = (MAX_N_NODES + 1) - 1;
    *queue_size = 0;
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    // Subtree above new_fixed_ind:
    if (father[new_fixed_ind] != 0) {
        up_to_fixed_ind(new_fixed_ind, status, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
        shortint current_root_ind = holder[(*holder_back - 1) % (MAX_N_NODES + 1)];
        // Prune unstretched nodes first:
        prune(current_root_ind, status, father, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_front, holder_back, holder_size);
        // Update stretched nodes:
        update_v_reborn_stretched(current_root_ind, v, gamma, status, father, num_kids, kids, seen, queue, queue_front, queue_back, queue_size, holder, holder_front, holder_back, holder_size);
        // Update unstretched nodes last:
        update_v_reborn_unstretched(current_root_ind, v, gamma, status, father, num_kids, kids, queue, queue_front, queue_back, queue_size);
    }
    // Subtrees below new_fixed_ind:
    for (shortint i = 0; i < num_kids[new_fixed_ind]; ++i) {
        shortint current_root_ind = kids[new_fixed_ind * (MAX_N_NODES + 1) + i];
        // !Prune unstretched nodes first!:
        prune(current_root_ind, status, father, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_front, holder_back, holder_size);
        // !Only update stretched nodes for if the current root is free!:
        if (status[current_root_ind] == 0) {
            update_v_reborn_stretched(current_root_ind, v, gamma, status, father, num_kids, kids, seen, queue, queue_front, queue_back, queue_size, holder, holder_front, holder_back, holder_size);
        }
        // Update unstretched nodes last:
        update_v_reborn_unstretched(current_root_ind, v, gamma, status, father, num_kids, kids, queue, queue_front, queue_back, queue_size);
    }
    v[0] = 0;
    v[new_fixed_ind] = 1;
}

// Update ddf, as well as several status-type things:
__host__ __device__ realnumber update_ddf_reborn(realnumber ddf, shortint new_fixed_ind, realnumber v[], realnumber v_old[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim, shortint queue[], shortint *queue_front, shortint *queue_back, shortint *queue_size, shortint holder[], shortint *holder_front, shortint *holder_back, shortint *holder_size, shortint seen[], realnumber gamma[]) {
    realnumber old_contribution = 0;
    realnumber new_contribution = 0;
    // Reset holder:
    *holder_front = 0;
    *holder_back = (MAX_N_NODES + 1) - 1;
    *holder_size = 0;
    // Find all nodes in the subtree above new_fixed_ind:
    shortint current_root_ind = new_fixed_ind;
    if (father[new_fixed_ind] == 0) {
        old_contribution += (v_old[new_fixed_ind] - 0) * (v_old[new_fixed_ind] - 0);
        new_contribution += (v[new_fixed_ind] - 0) * (v[new_fixed_ind] - 0);
    }
    else {
        up_to_fixed_ind(new_fixed_ind, status, father, num_kids, kids, holder, holder_front, holder_back, holder_size);
        current_root_ind = holder[(*holder_back - 1) % (MAX_N_NODES + 1)];
        // Look up as we are at current root:
        old_contribution += (v_old[current_root_ind] - v_old[father[current_root_ind]]) * (v_old[current_root_ind] - v_old[father[current_root_ind]]);
        new_contribution += (v[current_root_ind] - v[father[current_root_ind]]) * (v[current_root_ind] - v[father[current_root_ind]]);
        // Reset holder:
        *holder_front = 0;
        *holder_back = (MAX_N_NODES + 1) - 1;
        *holder_size = 0;
        // Find all nodes in above subtree:
        bfs_to_fixed_get_all_new(current_root_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
    }
    // Find all nodes in below subtrees:
    shortint current_num_kids = num_kids[new_fixed_ind];
    for (shortint i = 0; i < current_num_kids; ++i) {
        shortint current_kid_ind = kids[new_fixed_ind * (MAX_N_NODES + 1) + i];
        bfs_to_fixed_get_all_new(current_kid_ind, status, num_kids, kids, queue, queue_front, queue_back, queue_size, holder, holder_back, holder_size);
        // Look up as we are at current root:
        old_contribution += (v_old[current_kid_ind] - v_old[father[current_kid_ind]]) * (v_old[current_kid_ind] - v_old[father[current_kid_ind]]);
        new_contribution += (v[current_kid_ind] - v[father[current_kid_ind]]) * (v[current_kid_ind] - v[father[current_kid_ind]]);
    }
    // Now holder contains all other nodes needed
    while (*holder_size != 0) {
        shortint current_node_ind = holder[*holder_front];
        *holder_front = (*holder_front + 1) % (MAX_N_NODES + 1);
        --(*holder_size);
        // Now we only need to look down:
        shortint kids_num = num_kids[current_node_ind];
        for (shortint i = 0; i < kids_num; ++i) {
            old_contribution += (v_old[current_node_ind] - v_old[kids[current_node_ind * (MAX_N_NODES + 1) + i]]) * (v_old[current_node_ind] - v_old[kids[current_node_ind * (MAX_N_NODES + 1) + i]]);
            new_contribution += (v[current_node_ind] - v[kids[current_node_ind * (MAX_N_NODES + 1) + i]]) * (v[current_node_ind] - v[kids[current_node_ind * (MAX_N_NODES + 1) + i]]);
        }
        // Reset seen-status:
        seen[current_node_ind] = 0;
        // Reset status to be free:
        status[current_node_ind] = 0;
        // Reset gamma:
        gamma[current_node_ind] = 1;
    }
    //printf("(old, new) = (%f, %f)\n", old, new);
    return ddf + (new_contribution - old_contribution);
}

__host__ __device__ realnumber update_df(realnumber df, realnumber ddf, realnumber t, realnumber t_old) {
    return df + ddf * (t - t_old);
}

__host__ __device__ void best_tree_reborn(realnumber z[], realnumber value[], shortint status[], shortint father[], shortint num_kids[], shortint kids[], shortint dim) {
    // Define containers:
    shortint queue[(MAX_N_NODES + 1)];
    shortint queue_front = 0;
    shortint queue_back = (MAX_N_NODES + 1) - 1;
    shortint queue_size = 0;
    shortint holder[(MAX_N_NODES + 1)];
    shortint holder_front = 0;
    shortint holder_back = (MAX_N_NODES + 1) - 1;
    shortint holder_size = 0;
    // Define initial data:
    shortint level[(MAX_N_NODES + 1)];
    bfs_get_level_new(status, level, num_kids, kids, queue, &queue_front, &queue_back, &queue_size);
    shortint seen[(MAX_N_NODES + 1)] = {0};
    realnumber v[(MAX_N_NODES + 1)] = {0};
    realnumber v_old[(MAX_N_NODES + 1)] = {0};
    realnumber gamma[(MAX_N_NODES + 1)] = {1};
    realnumber max_heap[(MAX_N_NODES + 1)];
    shortint heap_to_tree_index[(MAX_N_NODES + 1)];
    shortint tree_to_heap_index[(MAX_N_NODES + 1)];
    for (shortint i = 0; i <= dim; ++i) {
        max_heap[i] = -INFINITY;
        heap_to_tree_index[i] = kids[0];
        tree_to_heap_index[i] = kids[0];
        gamma[i] = 1;
        z[i] = 0;
        v[i] = 0;
    }
    shortint heap_size = 0;
    // First turning point:
    realnumber n_max = value[1];
    shortint new_fixed_ind = 0, pre_fixed_ind = 0;
    for (shortint i = 1; i <= dim; ++i) {
        if (value[i] >= n_max) {
            n_max = value[i];
        }
    }
    shortint top_level = dim + 1;
    switch(dim) {
        case 7:
#pragma unroll 7
            for (shortint i = 1; i <= 7; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
            break;
        case 8:
#pragma unroll 8
            for (shortint i = 1; i <= 8; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
            break;
        case 9:
#pragma unroll 9
            for (shortint i = 1; i <= 9; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
            break;
        case 10:
#pragma unroll 10
            for (shortint i = 1; i <= 10; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
            break;
        case 11:
#pragma unroll 11
            for (shortint i = 1; i <= 11; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
            break;
        case 12:
#pragma unroll 12
            for (shortint i = 1; i <= 12; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
            break;
        default:
            for (shortint i = 1; i <= dim; ++i) {
                if (value[i] == n_max && level[i] < top_level) {
                    new_fixed_ind = i;
                    top_level = level[i]; // Fix the node with smallest level first
                }
            }
    }
    realnumber t = n_max, t_old = n_max;
    heap_insert(t, kids[0], max_heap, heap_to_tree_index, tree_to_heap_index, &heap_size);
    status[new_fixed_ind] = 2;
    seen[new_fixed_ind] = 1;
    update_v_reborn(new_fixed_ind, v, gamma, status, father, num_kids, kids, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size, seen);
    realnumber z_timestamp[MAX_N_NODES + 1];
    switch(dim) {
        case 7:
#pragma unroll 8
            for (shortint i = 0; i <= 7; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
            break;
        case 8:
#pragma unroll 9
            for (shortint i = 0; i <= 8; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
            break;
        case 9:
#pragma unroll 10
            for (shortint i = 0; i <= 9; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
            break;
        case 10:
#pragma unroll 11
            for (shortint i = 0; i <= 10; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
            break;
        case 11:
#pragma unroll 12
            for (shortint i = 0; i <= 11; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
            break;
        case 12:
#pragma unroll 13
            for (shortint i = 0; i <= 12; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
            break;
        default:
            for (shortint i = 0; i <= dim; ++i) {
                z_timestamp[i] = t;
                seen[i] = 0;
            }
    }
    seen[new_fixed_ind] = 1;
    z[new_fixed_ind] = z[new_fixed_ind] + (t - z_timestamp[new_fixed_ind]);
    z_timestamp[new_fixed_ind] = t;
    realnumber df = 0, df_old = 0, ddf = 0;
    // Look for df = -1:
    while (df > -1) {
        // Keep records:
        t_old = t;
        df_old = df;
        pre_fixed_ind = new_fixed_ind;
        // Find the next turning point:
        new_fixed_ind = next_turn_better_new(new_fixed_ind, t_old, z, v, value, status, father, num_kids, kids, dim, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size, max_heap, heap_to_tree_index, tree_to_heap_index, &heap_size);
        t = max_heap[0];
        //printf("Next fixed point: %f\n", t);
        if (t == -INFINITY) {
            // Compute the slop of the last segment before break:
            ddf = update_ddf_reborn(ddf, pre_fixed_ind, v, v_old, status, father, num_kids, kids, dim, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size, seen, gamma);
            break;
        }
        // Update ddf before update status:
        ddf = update_ddf_reborn(ddf, pre_fixed_ind, v, v_old, status, father, num_kids, kids, dim, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size, seen, gamma);
        // Update df:
        df = update_df(df, ddf, t, t_old);
        //printf("df = %f\n", df);
        // Update status:
        status[new_fixed_ind] = 2;
        seen[new_fixed_ind] = 1;
        // Only update these if we still haven't found df = -1:
        if (df > -1) {
            // Update z first:
            update_z_new(new_fixed_ind, value, t, z, z_timestamp, v, status, father, num_kids, kids, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size);
            // Update v_old:
            update_v_old_new(new_fixed_ind, pre_fixed_ind, t, v, v_old, status, father, num_kids, kids, dim, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size);
            // Update v:
            update_v_reborn(new_fixed_ind, v, gamma, status, father, num_kids, kids, queue, &queue_front, &queue_back, &queue_size, holder, &holder_front, &holder_back, &holder_size, seen);
        }
    }
    // Get t, z at df = -1:
    t = t_old - (df_old + 1) / ddf;
    z[0] = t;
    for (shortint i = 1; i <= dim; ++i) {
        z[i] = z[i] + v[i] * (t - z_timestamp[i]);
    }
}

//
//
//
//
//
//****************************** END OF BEI's NEW CODE ******************************
//
//
//










/*
 This function uses DFS to compute the father nodes of each node, and store the result in fathers_list[]:
 */
__host__ __device__ void dfs_tree_compute_fathers_non_recursive_array(shortint num_nodes,shortint *fathers_list, shortint *adj, shortint *deg, shortint root, shortint *stack, shortint *visited){
    fathers_list[0] = -1;
    shortint curr = root;
    shortint stack_depth = 0;
    stack[stack_depth] = curr;
    stack_depth = stack_depth + 1;
    fathers_list[root + 1] = 0;
    
    while(stack_depth > 0){
        stack_depth = stack_depth - 1;
        curr = stack[stack_depth];
        visited[curr] = 1 - visited[curr];
        for (shortint j = 0; j < deg[curr]; j++){
            shortint child_ix = adj[curr*num_nodes + j];
            if (visited[child_ix] == 1 - visited[root]){
                stack[stack_depth] = child_ix;
                stack_depth = stack_depth + 1;
                fathers_list[child_ix + 1] = curr + 1;
            }
        }
    }
}

/*
 This function uses DFS to compute ntilde from tree structure and input data:
 */
__host__ __device__ void dfs_tree_compute_ntilde_non_recursive_array(shortint num_nodes, shortint *adj, shortint *deg, shortint root, shortint *stack, shortint *visited, realnumber *ntilde, realnumber *data,  shortint T, shortint t){
    ntilde[0] = 0;
    shortint curr = root;
    shortint stack_depth = 0;
    stack[stack_depth] = curr;
    stack_depth = stack_depth + 1;
    ntilde[curr + 1] = data[curr*T + t];
    
    while(stack_depth > 0){
        stack_depth = stack_depth - 1;
        curr = stack[stack_depth];
        visited[curr] = 1 - visited[curr];
        for (shortint j = 0; j < deg[curr]; j++){
            shortint child_ix = adj[curr*num_nodes + j];
            if (visited[child_ix] == 1 - visited[root]){
                stack[stack_depth] = child_ix;
                stack_depth = stack_depth + 1;
                ntilde[child_ix + 1] = ntilde[curr + 1] + data[child_ix*T + t];
            }
        }
    }
}

/*
 This function converts tree structure data into the format needed for cost1 and cost2
 */
__host__ __device__ void convert_tree_data(shortint num_nodes, shortint *final_degrees, shortint *adj_list, shortint *father_list, shortint root_node, shortint num_kids[], shortint kids[]) {
    // Virtural node:
    num_kids[0] = 1;
    kids[0] = root_node + 1;
    // num_nodes:
    for (int i = 1; i <= num_nodes; ++i) {
        if (i == root_node + 1) {
            num_kids[i] = final_degrees[i - 1];
        }
        else {
            num_kids[i] = final_degrees[i - 1] - 1;
        }
    }
    // kids:
    
    switch(num_nodes) {
        case 7:
#pragma unroll 7
            for (shortint i = 1; i <= 7; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
            break;
        case 8:
#pragma unroll 8
            for (shortint i = 1; i <= 8; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
            break;
        case 9:
#pragma unroll 9
            for (shortint i = 1; i <= 9; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
            break;
        case 10:
#pragma unroll 10
            for (shortint i = 1; i <= 10; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
            break;
        case 11:
#pragma unroll 11
            for (shortint i = 1; i <= 11; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
            break;
        case 12:
#pragma unroll 12
            for (shortint i = 1; i <= 12; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
            break;
        default:
            for (shortint i = 1; i <= num_nodes; ++i) {
                if (i == root_node + 1) {
                    for (shortint j = 0; j <= num_kids[i]; ++j) {
                        // Add one extra spot to accomadate possible effective node
                        // Default to -2 indicating no effective node present
                        if (j == num_kids[i]) {
                            kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                        }
                        else {
                            kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        }
                    }
                }
                else {
                    shortint father_ind = father_list[i];
                    shortint j = 0;
                    while (j < num_kids[i] && adj_list[(i-1)*num_nodes + j] + 1 != father_ind) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j] + 1;
                        ++j;
                    }
                    while (j < num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + j] = adj_list[(i-1)*num_nodes + j + 1] + 1;
                        ++j;
                    }
                    // Add one extra spot to accomadate possible effective node
                    // Default to -2 indicating no effective node present
                    if (j == num_kids[i]) {
                        kids[i * (MAX_N_NODES + 1) + num_kids[i]] = -2;
                    }
                }
            }
    }
}


/*
 This function uses DFS to compute the entire cost of a given tree at a single time t
 */
__host__ __device__ realnumber dfs_tree_cost_from_z_non_recursive(shortint num_nodes, shortint *adj_list, shortint *final_degrees, shortint root_node, shortint *stack, shortint *visited, realnumber *z, realnumber *data, shortint T, shortint t) {
    
    shortint curr = root_node;
    shortint stack_depth = 0;
    stack[stack_depth] = curr;
    stack_depth = stack_depth + 1;
    // Root:
    realnumber temp2 = (z[curr + 1] + data[curr*T + t]);
    realnumber temp = (temp2*temp2);
    // Other nodes:
    while(stack_depth > 0){
        stack_depth = stack_depth - 1;
        curr = stack[stack_depth];
        visited[curr] = 1 - visited[curr];
        for (shortint j = 0; j < final_degrees[curr]; j++){
            shortint child_ix = adj_list[curr*num_nodes + j];
            if (visited[child_ix] == 1 - visited[root_node]){
                stack[stack_depth] = child_ix;
                stack_depth = stack_depth + 1;
                temp2 = ((z[child_ix + 1] - z[curr + 1]) + data[child_ix*T + t]);
                temp +=  (temp2*temp2);
            }
        }
    }
    return temp;
}

/*
 This function uses DFS to compute the entire cost of a given tree over all time t in [0, T)
 */
__host__ __device__ realnumber tree_cost_bei_array(shortint num_nodes, shortint T, realnumber *data, shortint root_node, edge *tree, shortint *adjacency_mat, shortint *final_degrees, shortint *adj_list){
    
    realnumber z[2*MAX_N_NODES + 1];
    realnumber ntilde[2*MAX_N_NODES + 1];
    shortint stack[MAX_N_NODES];
    shortint visited[MAX_N_NODES];
    shortint father_list[2*MAX_N_NODES + 1];
    shortint num_kids[MAX_N_NODES + 1];
    shortint kids[(MAX_N_NODES + 1) * (MAX_N_NODES + 1)];
    
    //float time_tree_cost_bei_array = 0;
    //float time_compute_f_i = 0;
    //float time_best_tree_array = 0;
    //float time_best_tree_array_avg = 0;
    //clock_t start_time = clock();
    
    // we need to set visited to be all zeros. We only need to do this once.
    // the rest does not need to be initialized
    for (shortint i = 0; i < num_nodes; i++){
        visited[i] = 0;
    }
    
    realnumber error_tree_model = 0.0;
    
    dfs_tree_compute_fathers_non_recursive_array(num_nodes, father_list, adj_list, final_degrees, root_node, stack, visited);
    
    convert_tree_data(num_nodes, final_degrees, adj_list, father_list, root_node, num_kids, kids); //maybe remove in the future
    
    //clock_t start_time2 = clock();
    // go over all the nodes and compute f_i - sum_children f_j
    for (shortint t = 0; t < T; ++t) {
        // Reset status of all nodes to be free for the next iteration:
        shortint status[2*MAX_N_NODES + 1] = {0};
        status[0] = 2;
        
        // we explore the tree using BFS to compute tiln = U'*F;
        dfs_tree_compute_ntilde_non_recursive_array(num_nodes, adj_list, final_degrees, root_node, stack, visited, ntilde, data, T, t);
        
        // Compute z_i for all i:
        //clock_t start_time3 = clock();
        //best_tree_array(ntilde, z, status, father_list, num_kids, kids, num_nodes);
        best_tree_reborn(z, ntilde, status, father_list, num_kids, kids, num_nodes);
        //clock_t end_time3 = clock();
        //time_best_tree_array = time_best_tree_array + (float) (end_time3 - start_time3) / CLOCKS_PER_SEC * 1000.0;
        
        // Compute square_norm((z_i - z_i_father) + F_i):
        realnumber tmp_cost = dfs_tree_cost_from_z_non_recursive(num_nodes, adj_list, final_degrees, root_node, stack, visited, z, data, T, t);
        
        error_tree_model += tmp_cost;
    }
    //clock_t end_time2 = clock();
    
    //clock_t end_time = clock();
    
    //time_best_tree_array_avg = time_best_tree_array / T;
    //printf( "time_best_tree_array_avg across T samples = %f\n", time_best_tree_array_avg );
    //time_compute_f_i = (float) (end_time2 - start_time2) / CLOCKS_PER_SEC * 1000.0;
    //printf( "time_compute_f_i = %f\n", time_compute_f_i );
    //time_tree_cost_bei_array = (float) (end_time - start_time) / CLOCKS_PER_SEC * 1000.0;
    //printf( "time_tree_cost_bei_array = %f\n", time_tree_cost_bei_array );
    return error_tree_model;
}

__host__ __device__ realnumber tree_cost_bei_array_inner(shortint num_nodes, shortint T, realnumber *data, shortint root_node, edge *tree, shortint *adjacency_mat, shortint *final_degrees, shortint *adj_list){
    
    realnumber z[2*MAX_N_NODES + 1];
    realnumber ntilde[2*MAX_N_NODES + 1];
    shortint stack[MAX_N_NODES];
    shortint visited[MAX_N_NODES];
    shortint father_list[2*MAX_N_NODES + 1];
    shortint num_kids[MAX_N_NODES + 1];
    shortint kids[(MAX_N_NODES + 1) * (MAX_N_NODES + 1)];
    
    // we need to set visited to be all zeros. We only need to do this once.
    // the rest does not need to be initialized
    for (shortint i = 0; i < num_nodes; i++){
        visited[i] = 0;
    }
    
    realnumber error_tree_model = 0.0;
    
    dfs_tree_compute_fathers_non_recursive_array(num_nodes, father_list, adj_list, final_degrees, root_node, stack, visited);
    
    convert_tree_data(num_nodes, final_degrees, adj_list, father_list, root_node, num_kids, kids); //maybe remove in the future
    
    // go over all the nodes and compute f_i - sum_children f_j
    for (shortint t = 0; t < T; ++t) {
        // Reset status of all nodes to be free for the next iteration:
        shortint status[2*MAX_N_NODES + 1] = {0};
        status[0] = 2;
        
        // we explore the tree using BFS to compute tiln = U'*F;
        dfs_tree_compute_ntilde_non_recursive_array(num_nodes, adj_list, final_degrees, root_node, stack, visited, ntilde, data, T, t);
        
        // Compute z_i for all i:
        best_tree_array_inner(ntilde, z, status, father_list, num_kids, kids, num_nodes);
        
        // Compute square_norm((z_i - z_i_father) + F_i):
        realnumber tmp_cost = dfs_tree_cost_from_z_non_recursive(num_nodes, adj_list, final_degrees, root_node, stack, visited, z, data, T, t);
        
        error_tree_model += tmp_cost;
    }
    
    return error_tree_model;
}

/*
 Update and maintain a max heap to store the smallest k costs and keep an array of corresponding tree indices:
 */
__host__ __device__ void heapify(realnumber cost, longint tree_index, realnumber max_heap[], longint smallest_trees[], shortint k) {
    if (cost >= max_heap[0]) {
        return;
    }
    else {
        // Remove the original max:
        realnumber temp = max_heap[k - 1];
        max_heap[0] = temp;
        longint temp_ind = smallest_trees[k - 1];
        smallest_trees[0] = temp_ind;
        shortint current_ind = 0;
        // Bubble down:
        while (1) {
            shortint left_ind = 2*current_ind + 1;
            shortint right_ind = 2*current_ind + 2;
            // Make sure we are reaching within boundary:
            if (left_ind < k && right_ind < k) {
                if (temp < max_heap[left_ind] && temp < max_heap[right_ind]) {
                    if (max_heap[left_ind] >= max_heap[right_ind]) {
                        max_heap[current_ind] = max_heap[left_ind];
                        max_heap[left_ind] = temp;
                        smallest_trees[current_ind] = smallest_trees[left_ind];
                        smallest_trees[left_ind] = temp_ind;
                        current_ind = left_ind;
                    }
                    else {
                        max_heap[current_ind] = max_heap[right_ind];
                        max_heap[right_ind] = temp;
                        smallest_trees[current_ind] = smallest_trees[right_ind];
                        smallest_trees[right_ind] = temp_ind;
                        current_ind = right_ind;
                    }
                }
                else if (temp < max_heap[left_ind] && temp >= max_heap[right_ind]) {
                    max_heap[current_ind] = max_heap[left_ind];
                    max_heap[left_ind] = temp;
                    smallest_trees[current_ind] = smallest_trees[left_ind];
                    smallest_trees[left_ind] = temp_ind;
                    current_ind = left_ind;
                }
                else if (temp < max_heap[right_ind] && temp >= max_heap[left_ind]) {
                    max_heap[current_ind] = max_heap[right_ind];
                    max_heap[right_ind] = temp;
                    smallest_trees[current_ind] = smallest_trees[right_ind];
                    smallest_trees[right_ind] = temp_ind;
                    current_ind = right_ind;
                }
                else {
                    break;
                }
            }
            else if (left_ind < k && temp < max_heap[left_ind]) {
                max_heap[current_ind] = max_heap[left_ind];
                max_heap[left_ind] = temp;
                smallest_trees[current_ind] = smallest_trees[left_ind];
                smallest_trees[left_ind] = temp_ind;
                current_ind = left_ind;
            }
            else if (right_ind < k && temp < max_heap[right_ind]) {
                max_heap[current_ind] = max_heap[right_ind];
                max_heap[right_ind] = temp;
                smallest_trees[current_ind] = smallest_trees[right_ind];
                smallest_trees[right_ind] = temp_ind;
                current_ind = right_ind;
            }
            else {
                break;
            }
        }
        // Put cost at the end of array:
        max_heap[k - 1] = cost;
        smallest_trees[k - 1] = tree_index;
        current_ind = k - 1;
        // Bubble up:
        while (1) {
            shortint father_ind = (current_ind - 1) / 2; // Integer division acts like floor function
            if (cost > max_heap[father_ind]) {
                max_heap[current_ind] = max_heap[father_ind];
                max_heap[father_ind] = cost;
                smallest_trees[current_ind] = smallest_trees[father_ind];
                smallest_trees[father_ind] = tree_index;
                current_ind = father_ind;
            }
            else {
                break;
            }
        }
    }
}

//
//
//
//
//
//****************************** END OF BEI's CODE ******************************
//
//
//












longint pow_int(shortint n, shortint p){
    longint r = 1;
    for (shortint i = 1; i <= p; i++){
        r = r*n;
    }
    return r;
}

// maybe this will have to be implemented in a non recursive manner
// depth-first-search to form the vector U for a given time slice
// the notion of "visited" is relative so that we do not need to reinitialize the vector visited each time we call bfs_tree
// the bfs_tree will induce a direction on the tree starting at the root node
__host__ __device__ void dfs_tree_cost(shortint num_nodes, shortint * adj, shortint * deg, shortint curr, shortint * visited, realnumber * U, realnumber * data,  shortint T, shortint t){
    
    visited[curr] = 1 - visited[curr]; // this marks the node as visited
    
    U[curr] = data[curr*T + t];
    
    for (shortint j = 0; j < deg[curr]; j++){
        shortint child_ix = adj[curr*num_nodes + j];
        if ( visited[child_ix] == 1 - visited[curr] ){ // we compare the current status with the status of the node we just visited
            
            U[curr] = U[curr] - data[child_ix*T + t];
            
            dfs_tree_cost(num_nodes, adj, deg, child_ix, visited, U, data,T , t);
        }
        
    }
    
    
}

// this is a non-recursive version of the depth-first-search algorithm
// maybe the GPU code will work better with non recursive algorithms
__host__ __device__ void dfs_tree_cost_non_recursive(shortint num_nodes, shortint * adj, shortint * deg, shortint root, shortint * stack, shortint * visited, realnumber * U, realnumber * data,  shortint T, shortint t){
    
    shortint curr = root;
    
    shortint stack_depth = 0;
    
    stack[stack_depth] = curr;
    stack_depth = stack_depth + 1;
    
    //printf("LIST: ");
    while(stack_depth > 0){
        
        stack_depth = stack_depth - 1;
        curr = stack[stack_depth];
        visited[curr] = 1 - visited[curr];
        
        //printf("(%d, %d) ", curr,deg[curr]);
        
        U[curr] = data[curr*T + t];
        
        for (shortint j = 0; j < deg[curr]; j++){
            shortint child_ix = adj[curr*num_nodes + j];
            //printf(":%d:%d ",child_ix,visited[child_ix]);
            if (visited[child_ix] == 1 - visited[root]){
                stack[stack_depth] = child_ix;
                stack_depth = stack_depth + 1;
                
                U[curr] = U[curr] - data[child_ix*T + t];
            }
        }
    }
    //printf("\n");
}


// this projects an element into the simplex where vectors are non-negative and must sum to one
// the algorithm runs in N unsigned N time
// the algorithm requires temporary arrays nsorted and cumsum. These arrays do not need to be initialized
// the vectors n and x need to be allocated but only n needs to be initialized
// if nooutput == 1 then we ignore x and simply ouput the error
// we return the distance squared because we will have to sum this distance with other distances when we are dealing with multiple time instants
__host__ __device__ realnumber projection_onto_simplex(shortint nooutput, realnumber *x, realnumber *n, shortint numvars, realnumber *nsorted, realnumber *cumsum){
    
    realnumber distance_squared = 0;
    
    for (shortint i = 0; i < numvars; i++){
        nsorted[i] = -n[ i ];
    }
    //sort
    sort_using_networks(nsorted, numvars);
    
    cumsum[0] = -nsorted[0];
    
    for (shortint i = 1; i < numvars; i++){
        cumsum[i] = (-nsorted[i]) + cumsum[i-1];
    }
    //find switch ix
    shortint ix = 0;
    
    realnumber maxval = 100000000;
    for (shortint i = 0; i < numvars; i++){
        realnumber tmp = (-nsorted[i]) + (1.0/ (1.0 + (realnumber) i))*(1 - cumsum[i]);
        
        if (tmp > 0 && tmp < maxval){
            maxval = tmp;
            ix = i;
        }
    }
    //find lambda
    
    realnumber lambda =  (1.0/ (1.0 + (realnumber) ix))*(1 - cumsum[ix]);
    
    //create final x
    for (shortint i = 0; i < numvars; i++){
        realnumber tmp = 	n[i] + lambda;
        
        realnumber xi;
        if (tmp<0){
            xi = 0;
        }else{
            xi = tmp;
        }
        
        distance_squared = distance_squared + (xi - n[i])*(xi - n[i]);
        
        // we might not be interested in the output in which we case we skip this part
        if (nooutput == 0){
            x[i] = xi;
        }
        
    }
    
    return distance_squared;
    
}

// this does the projection according to Condats' algorithm which is substantially faster than ours
__device__ __host__ realnumber projection_onto_simplex_condat(shortint nooutput, realnumber* x,  realnumber* y, const unsigned int length) {

	realnumber distance_squared = 0;
	const realnumber a = 1;    
	realnumber*    aux = x;
    realnumber*  aux0=aux;
    int        auxlength=1;
    int        auxlengthold=-1;
    realnumber    tau=(*aux=*y)-a;
    int     i=1;
    for (; i<length; i++)
        if (y[i]>tau) {
            if ((tau+=((aux[auxlength]=y[i])-tau)/(auxlength-auxlengthold))
                <=y[i]-a) {
                tau=y[i]-a;
                auxlengthold=auxlength-1;
            }
            auxlength++;
        }
    if (auxlengthold>=0) {
        auxlength-=++auxlengthold;
        aux+=auxlengthold;
        while (--auxlengthold>=0)
            if (aux0[auxlengthold]>tau)
                tau+=((*(--aux)=aux0[auxlengthold])-tau)/(++auxlength);
    }
    do {
        auxlengthold=auxlength-1;
        for (i=auxlength=0; i<=auxlengthold; i++)
            if (aux[i]>tau)
                aux[auxlength++]=aux[i];
            else
                tau+=(tau-aux[i])/(auxlengthold-i+auxlength);
    } while (auxlength<=auxlengthold);
    
	for (i=0; i<length; i++){
		realnumber xi = (y[i]>tau ? y[i]-tau : 0.0);

	 	distance_squared = distance_squared + (xi - y[i])*(xi - y[i]);

		if (nooutput == 0){
			x[i]=xi;		
		}
	    
	}
	
	return distance_squared;

}


// this is just like the function above by the restriction sum(x) == 1 is now replaced by sum(x) <= 1
__host__ __device__ realnumber projection_onto_inner_simplex(shortint nooutput, realnumber *x, realnumber *n, shortint numvars, realnumber *nsorted, realnumber *cumsum){
    
    realnumber distance_squared = 0;
    
    for (shortint i = 0; i < numvars; i++){
        nsorted[i] = -n[ i ];
    }
    //sort
    sort_using_networks(nsorted, numvars);
    
    cumsum[0] = -nsorted[0];
    
    for (shortint i = 1; i < numvars; i++){
        cumsum[i] = (-nsorted[i]) + cumsum[i-1];
    }
    //find switch ix
    shortint ix = 0;
    
    realnumber maxval = 100000000;
    for (shortint i = 0; i < numvars; i++){
        realnumber tmp = (-nsorted[i]) + (1.0/ (1.0 + (realnumber) i))*(1 - cumsum[i]);
        
        if (tmp > 0 && tmp < maxval){
            maxval = tmp;
            ix = i;
        }
    }
    //find lambda
    realnumber lambda =  (1.0/ (1.0 + (realnumber) ix))*(1 - cumsum[ix]);
    
    // threshold because we now have sum(x) <=1 instead of sum(x) == 1
    if (lambda > 0){
        lambda = 0;
    }
    
    //create final x
    for (shortint i = 0; i < numvars; i++){
        realnumber tmp = 	n[i] + lambda;
        
        realnumber xi;
        if (tmp<0){
            xi = 0;
        }else{
            xi = tmp;
        }
        
        distance_squared = distance_squared + (xi - n[i])*(xi - n[i]);
        
        // we might not be interested in the output in which we case we skip this part
        if (nooutput == 0){
            x[i] = xi;
        }
        
    }
    
    return distance_squared;
    
}





__device__ __host__ realnumber projection_onto_inner_simplex_condat(shortint nooutput, realnumber* x,  realnumber* y, const unsigned int length) {

	realnumber distance_squared = 0;
	const realnumber a = 1;    
	realnumber*    aux = x;
    realnumber*  aux0=aux;
    int        auxlength=1;
    int        auxlengthold=-1;
    realnumber    tau=(*aux=*y)-a;
    int     i=1;
    for (; i<length; i++)
        if (y[i]>tau) {
            if ((tau+=((aux[auxlength]=y[i])-tau)/(auxlength-auxlengthold))
                <=y[i]-a) {
                tau=y[i]-a;
                auxlengthold=auxlength-1;
            }
            auxlength++;
        }
    if (auxlengthold>=0) {
        auxlength-=++auxlengthold;
        aux+=auxlengthold;
        while (--auxlengthold>=0)
            if (aux0[auxlengthold]>tau)
                tau+=((*(--aux)=aux0[auxlengthold])-tau)/(++auxlength);
    }
    do {
        auxlengthold=auxlength-1;
        for (i=auxlength=0; i<=auxlengthold; i++)
            if (aux[i]>tau)
                aux[auxlength++]=aux[i];
            else
                tau+=(tau-aux[i])/(auxlengthold-i+auxlength);
    } while (auxlength<=auxlengthold);
    
	if (tau < 0){
		tau = 0;	
	}

	for (i=0; i<length; i++){
		realnumber xi = (y[i]>tau ? y[i]-tau : 0.0);

	 	distance_squared = distance_squared + (xi - y[i])*(xi - y[i]);

		if (nooutput == 0){
			x[i]=xi;		
		}
	    
	}
	
	return distance_squared;

}



// here we will write several functions to evaluate the quality of the trees being generated
// all the functrions receive a liset of edges, an adjacency matrix, a list of degrees and an adjacancy list
// however, they might not make use of all of these
// the functions also receive an array that contains, per row, the evolution of a mutate position aunsigned time
// the function might make use of a scrape memory that needs to be initialized before hand, appropriately
// data is a matrix with num_nodes rows and T columns
__host__ __device__ realnumber tree_cost(shortint num_nodes, shortint T, realnumber * data, shortint root_node, edge *tree, shortint *adjacency_mat , shortint * final_degrees, shortint *adj_list){
    
    // extract from the scrap memory the different components we want to make use of
    //realnumber *nsorted = (realnumber *) scrapmem;
    //realnumber *cumsum = &(nsorted[num_nodes]);
    //realnumber *U_vec = &(cumsum[num_nodes]);
    //shortint *visited = (shortint *) &(U_vec[num_nodes]);
    //shortint *stack = (shortint *) &(visited[num_nodes]);
    
    realnumber nsorted[MAX_N_NODES];
    //realnumber cumsum[MAX_N_NODES];
    realnumber U_vec[MAX_N_NODES];
    shortint visited[MAX_N_NODES];
    shortint stack[MAX_N_NODES];
    
    // we need to set visited to be all zeros. We only need to do this once.
    // the rest does not need to be initialized
    for (int i = 0; i < num_nodes; i++){
        visited[i] = 0;
    }
    
    realnumber error_tree_model = 0;
    
    // go over all the nodes and compute f_i - sum_children f_j
    for (shortint t = 0; t < T; t++){
        
        // we explore the tree using BFS
        //dfs_tree_cost(num_nodes, adj_list, final_degrees, root_node, visited, U_vec, data,  T, t);
        dfs_tree_cost_non_recursive(num_nodes,adj_list, final_degrees,  root_node, stack, visited, U_vec, data,   T,  t);
        //printf(" U VECT\n");
        
        //for (int i = 0; i < num_nodes ; i++){
        //    printf("%f ",U_vec[i]);
        //}
        //printf("\n");
        
        
        // here the variable U has, for every point in time, a measure of the error in the tree model
        // note that it is important for the projection funtion to return the distance squared so that we can add these up
        //error_tree_model = error_tree_model + projection_onto_simplex(1, NULL, U_vec, num_nodes, nsorted, cumsum);
        error_tree_model = error_tree_model + projection_onto_simplex_condat(1, nsorted, U_vec, num_nodes);        

    }
    
    return error_tree_model;
}

__host__ __device__ realnumber tree_cost_inner(shortint num_nodes, shortint T, realnumber * data, shortint root_node, edge *tree, shortint *adjacency_mat , shortint * final_degrees, shortint *adj_list){
    
    // extract from the scrap memory the different components we want to make use of
    //realnumber *nsorted = (realnumber *) scrapmem;
    //realnumber *cumsum = &(nsorted[num_nodes]);
    //realnumber *U_vec = &(cumsum[num_nodes]);
    //shortint *visited = (shortint *) &(U_vec[num_nodes]);
    //shortint *stack = (shortint *) &(visited[num_nodes]);
    
    realnumber nsorted[MAX_N_NODES];
    //realnumber cumsum[MAX_N_NODES];
    realnumber U_vec[MAX_N_NODES];
    shortint visited[MAX_N_NODES];
    shortint stack[MAX_N_NODES];
    
    // we need to set visited to be all zeros. We only need to do this once.
    // the rest does not need to be initialized
    for (int i = 0; i < num_nodes; i++){
        visited[i] = 0;
    }
    
    realnumber error_tree_model = 0;
    
    // go over all the nodes and compute f_i - sum_children f_j
    for (shortint t = 0; t < T; t++){
        
        // we explore the tree using BFS
        //dfs_tree_cost(num_nodes, adj_list, final_degrees, root_node, visited, U_vec, data,  T, t);
        dfs_tree_cost_non_recursive(num_nodes,adj_list, final_degrees,  root_node, stack, visited, U_vec, data,   T,  t);
        //printf(" U VECT\n");
        
        //for (int i = 0; i < num_nodes ; i++){
        //    printf("%f ",U_vec[i]);
        //}
        //printf("\n");
        
        
        // here the variable U has, for every point in time, a measure of the error in the tree model
        // note that it is important for the projection funtion to return the distance squared so that we can add these up
        //error_tree_model = error_tree_model + projection_onto_inner_simplex(1, NULL, U_vec, num_nodes, nsorted, cumsum);
		error_tree_model = error_tree_model + projection_onto_inner_simplex_condat(1, nsorted, U_vec, num_nodes);
        
    }
    
    return error_tree_model;
}


// The perm has size n-2
// Each element in perm has numbers between 0 and num_tree_vertices - 1
__host__ __device__ void permutation(shortint num_tree_vertices, longint perm_index, shortint *perm)
{
    for (shortint k = num_tree_vertices - 2; k >= 1; --k )
    {
        perm[k - 1] = perm_index % num_tree_vertices;
        perm_index = perm_index / num_tree_vertices;
    }
}

// Degrees is a scrap memory position. In the end it will not contain anything useful
// The degrees arraw does not need to be initialized
// We assume that the smallest element of "code" starts at 0
// note that none of the arrays needs to be pre-initialized to any values. only the memory needs to be available

__host__ __device__ void prufer_tree(edge *tree, shortint *degrees, shortint *code, shortint num_tree_vertices)
{
    shortint i, j;
    
    shortint last_edge_ix = 0; // this is next free index to use for edges
    
    shortint code_len = num_tree_vertices - 2;
    
    
    // Start with 1 in all of them
    for (i = 0; i < num_tree_vertices; i++) {
        degrees[i] = 1;
    }
    
    // Add to the degrees the number of occurrences of each node index in the code
    for (i = 0; i < code_len; i++) {
        degrees[code[i]] = degrees[code[i]] + 1;
    }
    
    // Add edges to nodes in the code
    for (i = 0; i < code_len; i++) {
        // Find the lowest-numbered node with degree 1
        // note that the following codes requires j to be global withing this function
        for (j = 0; degrees[j] != 1; j++);
        
        // Add the edge
        tree[last_edge_ix].first = j; // Note that these indices start at zero
        tree[last_edge_ix].second = code[i]; // Note that these indices start at zero
        
        degrees[j] = degrees[j] - 1;
        degrees[code[i]] = degrees[code[i]] - 1;
        last_edge_ix = last_edge_ix + 1;
    }
    /* Find the last 2 degree-1 nodes */
    for (i = 0; degrees[i] != 1; i++);
    for (j = i + 1; degrees[j] != 1; j++);
    
    /* Add the last edge */
    tree[last_edge_ix].first = i;
    tree[last_edge_ix].second = j;
}

// the memory "integer_array" will never change
// degrees is a scrap memory position. in the end it will not contain anything useful
// regarding this function the permutation_array will also be scrap memory
// note that none of the arrays needs to be pre-initialized to any values. only the memory needs to be available
// adapted from the Martin Broadhurst method
__host__ __device__ void generate_tree_from_index(shortint num_tree_vertices, longint tree_index, shortint *permutation_array,  shortint *degrees, edge *tree, shortint *adjacency_mat , shortint * final_degrees, shortint *adj_list)
{
    // General purpose counters
    shortint j = 0;
    
    shortint source; //source node for an edge
    shortint destination;
    
    // Getting permutation from function
    permutation( num_tree_vertices, tree_index, permutation_array );
    
    // Function to provide Prufer_code tree structure
    prufer_tree( tree, degrees, permutation_array, num_tree_vertices );
    
    // Filling up the adjacency matrix
    // Note that a tree with   num_tree_vertices    vertices has (num_tree_vertices - 1) edges
    
    for (j = 0; j < num_tree_vertices ; j++){
        final_degrees[ j ] = 0;
    }
    for (j = 0; j < num_tree_vertices*num_tree_vertices ; j++){
        adjacency_mat[ j ] = 0;
    }
    switch(num_tree_vertices) {
        case 7:
#pragma unroll 6
            for ( j = 0; j < 6; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
            break;
        case 8:
#pragma unroll 7
            for ( j = 0; j < 7; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
            break;
        case 9:
#pragma unroll 8
            for ( j = 0; j < 8; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
            break;
        case 10:
#pragma unroll 9
            for ( j = 0; j < 9; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
            break;
        case 11:
#pragma unroll 10
            for ( j = 0; j < 10; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
            break;
        case 12:
#pragma unroll 11
            for ( j = 0; j < 11; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
            break;
        default:
            for ( j = 0; j < num_tree_vertices - 1; j++ ) {
                source = tree[j].first;
                destination = tree[j].second;
                // use 1D representation for the adjacency matrix
                adjacency_mat[   num_tree_vertices*source   +   destination] = 1;
                adjacency_mat[   num_tree_vertices*destination   +   source] = 1;
                // use a 1D representation for hte adjacency list that is over initialized since we have enough space
                adj_list[source*num_tree_vertices + final_degrees[source]] = destination;
                adj_list[destination*num_tree_vertices + final_degrees[destination]] = source;
                final_degrees[source] = final_degrees[source] + 1;
                final_degrees[destination] = final_degrees[destination] + 1;
                
            }
    }
}

// CUDA kernels with top k, one for each cost fucntion:
__global__ void kernel_to_compute_optimal_tree_only_global_mem_1k(int chunck_per_cycle, shortint num_devices, shortint device_index, longint total_num_trees, shortint num_tree_vertices, shortint  T, shortint root_node, realnumber *data, void *scrape_memory, shortint shift_size, realnumber *max_heap_device, longint *smallest_trees_device, int k){
    
    int baseix = blockIdx.x*blockDim.x + threadIdx.x;

    char * shifted_pointer = (char *) scrape_memory;
    shifted_pointer = shifted_pointer + shift_size*baseix;
    
    // Allocate top k stuff:
    realnumber max_heap[topk];
    longint smallest_trees[topk];
    for (int i = 0; i < topk; ++i) {
        max_heap[i] = topk * MAX_N_NODES + i;
        smallest_trees[i] = 0;
    }
    
    // we only use global memory here
    shortint * adjacency_mat = (shortint *) shifted_pointer;
    shortint * adjacency_list = (shortint *) &(adjacency_mat[num_tree_vertices*num_tree_vertices]);
    edge *tree = (edge *) &(adjacency_list[num_tree_vertices*num_tree_vertices]);
    shortint * final_degrees = (shortint *) &(tree[num_tree_vertices-1]);
    shortint * perm_scrape = (shortint*) &(final_degrees[num_tree_vertices-2]);
    shortint * deg_scrape = (shortint*) &(perm_scrape[num_tree_vertices]);
    
    for (long int ix = device_index  +  baseix*num_devices  ; ix < total_num_trees ; ix = ix + num_devices*chunck_per_cycle){
        
        generate_tree_from_index( num_tree_vertices, ix, perm_scrape, deg_scrape, tree, adjacency_mat , final_degrees, adjacency_list );
        
        realnumber val  = tree_prior(num_tree_vertices, adjacency_mat, adjacency_list, final_degrees); //here we include a prior cost on the tree  
		// if the prior says that the tree will topology is not allowed, we do not even compute the rest 			
		if (val < 10000){
			val +=  cost_trans_function(tree_cost_bei_array(num_tree_vertices, T, data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));			
		}

        // Push into max_heap:
        if (val == val && val < max_heap[0]) { // Avoid NaN
            heapify(val, ix, max_heap, smallest_trees, k);
        }
        
    }
    
    // Write top k stuff to global memory:
    for (shortint i = 0; i < k; ++i) {
        max_heap_device[baseix + i * chunck_per_cycle] = max_heap[i];
        smallest_trees_device[baseix + i * chunck_per_cycle] = smallest_trees[i];
    }
}

__global__ void kernel_to_compute_optimal_tree_only_global_mem_2k(int chunck_per_cycle, shortint num_devices, shortint device_index, longint total_num_trees, shortint num_tree_vertices, shortint  T, shortint root_node, realnumber *data, void *scrape_memory , shortint shift_size, realnumber *max_heap_device, longint *smallest_trees_device, int k){
    
    int baseix = blockIdx.x*blockDim.x + threadIdx.x;
    
    char * shifted_pointer = (char *) scrape_memory;
    shifted_pointer = shifted_pointer + shift_size*baseix;
    
    // Allocate top k stuff:
    realnumber max_heap[topk];
    longint smallest_trees[topk];
    for (int i = 0; i < topk; ++i) {
        max_heap[i] = topk * MAX_N_NODES + i;
        smallest_trees[i] = 0;
    }
    
    // we only use global memory here
    shortint * adjacency_mat = (shortint *) shifted_pointer;
    shortint * adjacency_list = (shortint *) &(adjacency_mat[num_tree_vertices*num_tree_vertices]);
    edge *tree = (edge *) &(adjacency_list[num_tree_vertices*num_tree_vertices]);
    shortint * final_degrees = (shortint *) &(tree[num_tree_vertices-1]);
    shortint * perm_scrape = (shortint*) &(final_degrees[num_tree_vertices-2]);
    shortint * deg_scrape = (shortint*) &(perm_scrape[num_tree_vertices]);
    
    for (long int ix = device_index  +  baseix*num_devices  ; ix < total_num_trees ; ix = ix + num_devices*chunck_per_cycle){
            
        generate_tree_from_index( num_tree_vertices, ix, perm_scrape, deg_scrape, tree, adjacency_mat , final_degrees, adjacency_list );
        
		realnumber val  = tree_prior(num_tree_vertices, adjacency_mat, adjacency_list, final_degrees); //here we include a prior cost on the tree  
		// if the prior says that the tree will topology is not allowed, we do not even compute the rest 			
		if (val < 10000){
			val +=  cost_trans_function(tree_cost_bei_array_inner(num_tree_vertices, T, data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));		
		}

        // Push into max_heap:
        if (val == val && val < max_heap[0]) { // Avoid NaN
            heapify(val, ix, max_heap, smallest_trees, k);
        }

    }
    
    // Write top k stuff to global memory:
    for (shortint i = 0; i < k; ++i) {
        max_heap_device[baseix + i * chunck_per_cycle] = max_heap[i];
        smallest_trees_device[baseix + i * chunck_per_cycle] = smallest_trees[i];
    }
}

__global__ void kernel_to_compute_optimal_tree_only_global_mem_3k(int chunck_per_cycle,shortint num_devices, shortint device_index,  longint total_num_trees, shortint num_tree_vertices, shortint  T, shortint root_node, realnumber *data, void *scrape_memory , shortint shift_size, realnumber *max_heap_device, longint *smallest_trees_device, int k){
    
    int baseix = blockIdx.x*blockDim.x + threadIdx.x;
    
    char * shifted_pointer = (char *) scrape_memory;
    shifted_pointer = shifted_pointer + shift_size*baseix;
    
    // Allocate top k stuff:
    realnumber max_heap[topk];
    longint smallest_trees[topk];
    for (int i = 0; i < topk; ++i) {
        max_heap[i] = topk * MAX_N_NODES + i;
        smallest_trees[i] = 0;
    }
    
    // we only use global memory here
    shortint * adjacency_mat = (shortint *) shifted_pointer;
    shortint * adjacency_list = (shortint *) &(adjacency_mat[num_tree_vertices*num_tree_vertices]);
    edge *tree = (edge *) &(adjacency_list[num_tree_vertices*num_tree_vertices]);
    shortint * final_degrees = (shortint *) &(tree[num_tree_vertices-1]);
    shortint * perm_scrape = (shortint*) &(final_degrees[num_tree_vertices-2]);
    shortint * deg_scrape = (shortint*) &(perm_scrape[num_tree_vertices]);
    
    for (long int ix = device_index  +  baseix*num_devices  ; ix < total_num_trees ; ix = ix + num_devices*chunck_per_cycle){
            
        generate_tree_from_index( num_tree_vertices, ix, perm_scrape, deg_scrape, tree, adjacency_mat , final_degrees, adjacency_list );
        
		realnumber val  = tree_prior(num_tree_vertices, adjacency_mat, adjacency_list, final_degrees); //here we include a prior cost on the tree  
		// if the prior says that the tree will topology is not allowed, we do not even compute the rest 			
		if (val < 10000){
			val += cost_trans_function(tree_cost(num_tree_vertices, T, data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));	
		}

        // Push into max_heap:
        if (val == val && val < max_heap[0]) { // Avoid NaN
            heapify(val, ix, max_heap, smallest_trees, k);
        }

    }
    
    // Write top k stuff to global memory:
    for (shortint i = 0; i < k; ++i) {
        max_heap_device[baseix + i * chunck_per_cycle] = max_heap[i];
        smallest_trees_device[baseix + i * chunck_per_cycle] = smallest_trees[i];
    }
}

__global__ void kernel_to_compute_optimal_tree_only_global_mem_4k(int chunck_per_cycle, shortint num_devices, shortint device_index, longint total_num_trees, shortint num_tree_vertices, shortint  T, shortint root_node, realnumber *data, void *scrape_memory , shortint shift_size, realnumber *max_heap_device, longint *smallest_trees_device, int k){
    
    int baseix = blockIdx.x*blockDim.x + threadIdx.x;
    
    char * shifted_pointer = (char *) scrape_memory;
    shifted_pointer = shifted_pointer + shift_size*baseix;
    
    // Allocate top k stuff:
    realnumber max_heap[topk];
    longint smallest_trees[topk];
    for (int i = 0; i < topk; ++i) {
        max_heap[i] = topk * MAX_N_NODES + i;
        smallest_trees[i] = 0;
    }
    
    // we only use global memory here
    shortint * adjacency_mat = (shortint *) shifted_pointer;
    shortint * adjacency_list = (shortint *) &(adjacency_mat[num_tree_vertices*num_tree_vertices]);
    edge *tree = (edge *) &(adjacency_list[num_tree_vertices*num_tree_vertices]);
    shortint * final_degrees = (shortint *) &(tree[num_tree_vertices-1]);
    shortint * perm_scrape = (shortint*) &(final_degrees[num_tree_vertices-2]);
    shortint * deg_scrape = (shortint*) &(perm_scrape[num_tree_vertices]);
    
    for (long int ix = device_index  +  baseix*num_devices  ; ix < total_num_trees ; ix = ix + num_devices*chunck_per_cycle){
            
        generate_tree_from_index( num_tree_vertices, ix, perm_scrape, deg_scrape, tree, adjacency_mat , final_degrees, adjacency_list );
        
		realnumber val  = tree_prior(num_tree_vertices, adjacency_mat, adjacency_list, final_degrees); //here we include a prior cost on the tree  
		// if the prior says that the tree will topology is not allowed, we do not even compute the rest 			
		if (val < 10000){
			val += cost_trans_function(tree_cost_inner(num_tree_vertices, T, data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));	
		}

        // Push into max_heap:
        if (val == val && val < max_heap[0]) { // Avoid NaN
            heapify(val, ix, max_heap, smallest_trees, k);
        }

    }
    
    // Write top k stuff to global memory:
    for (shortint i = 0; i < k; ++i) {
        max_heap_device[baseix + i * chunck_per_cycle] = max_heap[i];
        smallest_trees_device[baseix + i * chunck_per_cycle] = smallest_trees[i];
    }
}


//*********************************   TESTS


void test_projection(){


	realnumber input[] = {0.01, 0.01266, 0.0035,0.02,-1.3,-1.3,0.165,0.4,0.1,0.01,0.1,0.005};
	int num_nodes = 12;

	realnumber output[num_nodes];
	realnumber output_condat[num_nodes];
	realnumber nsorted[num_nodes];
	realnumber cumsum[num_nodes];

	int i = 0;	
	int j = 0;
	
	realnumber error_proj_simplex_our = 0;
	realnumber error_proj_simplex_condat = 0;

	//call functions
	float time;
	cstart();
	for (int i = 1; i < 100000; i++)
		error_proj_simplex_our = projection_onto_inner_simplex( 1, NULL, input, num_nodes, nsorted, cumsum );
	cend(&time);

	printf( "error_proj_simplex our_version = %f, totaltime %f \n", error_proj_simplex_our, time );
	for ( i = 0; i < num_nodes; i++ )
	{
		
		printf( "%f ", output[i] );
	}
 	printf( "\n" );

	cstart();
	for (int i = 1; i < 100000; i++)
		error_proj_simplex_condat = projection_onto_inner_simplex_condat( 1, nsorted, input, num_nodes );
	cend(&time);


	printf( "error_proj_simplex condat_version = %f, totaltime %f\n", error_proj_simplex_condat, time );
	for ( i = 0; i < num_nodes; i++ )
	{
		
		printf( "%f ", output_condat[i] );
	}
 	printf( "\n" );

	// compare
	int err = 0;
	for ( j = 0; j < num_nodes; j++ )
	{
		
		if ( output[j] != output_condat[j] )
		{
			err = 1;
		}
		
	}
	printf( "Error flag = %d\n", err );

}


//******************************************




int main(int argc, const char * argv[]) {


    // argv[1] = cpu, cpu_multithread, gpu
    // argv[2] = cost1 (bei), cost2 (bei_inner), cost3 (ray), cost4 (ray_inner)
    // argv[3] = num_nods
    // argv[4] = T
    // argv[5] = input_file
    // argv[6] = output_best_k
    // argv[7] = k
    // argv[8] = user_GPU_choice
    // argv[9] = number of CPU cores to use (might not be respected, the number input will be fixed to be a divisor of the number of trees)
    // argv[10] = number of devices that we will be using
    // argv[11] = which particular subset of trees the current device will work on
    // argv[12] = number of thread per block in CUDA
    // argv[13] = number of blocks
    

    
    // check the number of arguments
    if (!(argc == 12 || argc == 14)) {
        printf("Invalid number of inputs! Number = %d but should be 12 or 14. \n",argc);
        return 0;
    }
  
    
    // if number of arguments is OK, continue
    srand (time(NULL));
    
    // Get user input of device choice:
    char device_choice = 'C';
    if (strcmp(argv[1], "cpu") == 0) {
        device_choice = 'C';
    }
    else if (strcmp(argv[1], "cpu_multithread") == 0) {
        device_choice = 'M';
    }
    else if (strcmp(argv[1], "gpu") == 0) {
        device_choice = 'G';
    }
    else {
        printf("Invalid choice of device!\n");
        return 0;
    }
    
    // Get user input of cost function choice:
    char cost_choice = '1';
    if (strcmp(argv[2], "cost1") == 0) {
        cost_choice = '1';
    }
    else if (strcmp(argv[2], "cost2") == 0) {
        cost_choice = '2';
    }
    else if (strcmp(argv[2], "cost3") == 0) {
        cost_choice = '3';
    }
    else if (strcmp(argv[2], "cost4") == 0) {
        cost_choice = '4';
    }
    else {
        printf("Invalid choice of cost!\n");
        return 0;
    }
    
    FILE *input_file = fopen(argv[5], "r");
    FILE *output_best_k = fopen(argv[6], "w+");
    //FILE *run_time = fopen("run_time.txt", "w+");
    
    // Read input data:
    realnumber input_data[50000];
    int data_count = 0;
    fscanf(input_file, "%f", &input_data[data_count]);
    realnumber x = ((realnumber)rand() / (realnumber)(RAND_MAX/1e-4)) - 5e-5; // Random float between -5e-5 and 5e-5
    input_data[data_count] += x; // Add small noise to data
    while (!feof(input_file)) {
        ++data_count;
        fscanf(input_file, "%f", &input_data[data_count]);
        x = ((realnumber)rand() / (realnumber)(RAND_MAX/1e-4)) - 5e-5; // Random float between -5e-5 and 5e-5
        input_data[data_count] += x; // Add small noise to data
    }
    
    float cpu_time;
    float gpu_time;
    
    shortint num_tree_vertices = atoi(argv[3]);
    shortint root_node = 0;  // in the end of the day we are looking for rooted trees so we need to know which node is the root. Normally, a virtual node that represents the background and wildtype mutant
    longint total_num_trees = pow_int(num_tree_vertices, num_tree_vertices - 2);
    printf("Total number of trees: %lu\n", total_num_trees);
    
    //realnumber smallest_cost = 100000;
    
    // Create an array to act as max heap
    const int k = atoi(argv[7]);
    if (k > total_num_trees) {
        printf("k is too large!\n");
        return 0;
    }
    
    realnumber max_heap[k];
    // Create an array to store trees with smallest costs:
    longint smallest_trees[k];
    for (shortint i = 0; i < k; ++i) {
        max_heap[i] = 100 * MAX_N_NODES - i;
        smallest_trees[i] = 0;
    }
    
    shortint num_threads_for_openmp = 1;    //it is important that this is a divisor of the total number of trees which is n^(n-2)
    shortint num_cpu_core = atoi(argv[9]);

    // the number of threads can be large or smaller than the number of trees and does not have to be a multiple of the total number of trees
    // we are using an interleaved distribution of tree indices per device and per thread.
    num_threads_for_openmp = num_cpu_core;
    
    
    printf("Number of OpenMP threads: %d\n", num_threads_for_openmp);
    omp_set_num_threads(num_threads_for_openmp);
    
    // we are going to allow the current program being lauched to just process a subset of the total number of trees so that we can exploit multiple devices,  GPUs, CPUs, etc.
    shortint num_devices = atoi(argv[10]);
    shortint device_index = atoi(argv[11]) - 1;


    shortint T = atoi(argv[4]);
    longint tree_index = 0;
    
    // Create space for the Prufer code and also for the scrape degree array
    shortint code_length = num_tree_vertices - 2;
    shortint *permutation_array = (shortint *) malloc( code_length * sizeof(shortint) );
    shortint *degrees = (shortint *) malloc( num_tree_vertices * sizeof(shortint) );
    
    // Create space for the list of edges in the treee
    edge *tree = (edge*) malloc((num_tree_vertices - 1) * sizeof(edge));
    
    // Create space for the adjacency matrix
    shortint* adjacency_mat = (shortint *) malloc(num_tree_vertices*num_tree_vertices*sizeof(shortint));
    
    // Create space for the adjacency list
    shortint *final_degrees = (shortint *) malloc( num_tree_vertices * sizeof(shortint) );
    shortint* adjacency_list = (shortint *) malloc(num_tree_vertices*num_tree_vertices*sizeof(shortint)); // we are over initializing

    // Set regularization parameters for Path Algorithm 
    const shortint n_reg_params = 20;
    const realnumber rho_min=0, rho_max=1;
    realnumber *regularization_parameters = (realnumber * ) malloc(n_reg_params);
    for (int x = 0; x < n_reg_params; ++x) regularization_parameters[x] = x * (realnumber)((rho_max - rho_min) / n_reg_params); // linearly spaced regularization values to start??

    // Allocate arrays for top k trees for all OpenMP thread:
    realnumber *max_heap_mp = (realnumber *) malloc(k * num_threads_for_openmp * sizeof(realnumber));
    longint *smallest_trees_mp = (longint *) malloc(k * num_threads_for_openmp * sizeof(longint));
    // Allocate for regularization step 
    realnumber max_heap_with_regularization_mp[num_threads_for_openmp][n_reg_params * k];
    longint smallest_trees_with_regularization_mp[num_threads_for_openmp][n_reg_params * k];

    for (shortint i = 0; i < k * num_threads_for_openmp; ++i) {
        max_heap_mp[i] = 100 * k * num_threads_for_openmp - i;
        smallest_trees_mp[i] = 0;
    }
    
    if (device_choice == 'C' || device_choice == 'M') {
        if (device_choice == 'C') {
            // ************************* START OF SINGLE CORE CODE **********************************
			printf( "Single CPU core!\n" );
			
            cstart();
            for (tree_index = device_index; tree_index < total_num_trees ; tree_index = tree_index + num_devices){
                // Master function contains methods to generate permutations from index and generate prufer trees
                generate_tree_from_index(num_tree_vertices, tree_index, permutation_array, degrees, tree, adjacency_mat, final_degrees, adjacency_list);
                
                realnumber cost = 0;
                cost  = tree_prior(num_tree_vertices, adjacency_mat, adjacency_list, final_degrees); //here we include a prior cost on the tree  
                if (cost < 10000){
                    switch (cost_choice) {
                        case '1':
                            cost += cost_trans_function(tree_cost_bei_array(num_tree_vertices, T, input_data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));
                            break;
                        case '2':
                            cost += cost_trans_function(tree_cost_bei_array_inner(num_tree_vertices, T, input_data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));
                            break;
                        case '3':
                            cost += cost_trans_function(tree_cost(num_tree_vertices, T, input_data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));
                            break;
                        case '4':
                            cost += cost_trans_function(tree_cost_inner(num_tree_vertices, T, input_data, root_node, tree, adjacency_mat, final_degrees, adjacency_list));
                            break;
                        default:
                            printf("Invalid cost choice!\n");
                            break;
                    }
                }
                
                if (cost == cost && cost < max_heap[0]) { // Avoid NaN
                    heapify(cost, tree_index, max_heap, smallest_trees, k);
                }
            }
            cend(&cpu_time);
        }
        else if (device_choice == 'M') {

            // ************************* START OF MULTI CORE CODE **********************************
            realnumber costs[n_reg_params];
            memset(costs, 0, sizeof(realnumber) * n_reg_params);
            
            realnumber start = omp_get_wtime();
            #pragma omp parallel for //schedule(static)
            for (int part_num = 0; part_num < num_threads_for_openmp; part_num++){
                // we define severel local objects that need to be created for each of the parallel threads
                shortint local_permutation_array[MAX_N_NODES];
                shortint local_degrees[MAX_N_NODES];
                edge local_tree[MAX_N_NODES];
                shortint local_adjacency_mat[MAX_N_NODES*MAX_N_NODES];
                shortint local_final_degrees[MAX_N_NODES];
                shortint local_adjacency_list[MAX_N_NODES*MAX_N_NODES];
                
                // Top k stuff for each thread:
                realnumber max_heap_curr[k];
                realnumber list_of_maxheaps_foreach_regularization_param[n_reg_params][k];
                longint smallest_trees_curr[k];
                longint list_of_smallest_trees_foreach_regularization_param[n_reg_params][k];
                for (shortint i = 0; i < k; ++i) {
                    max_heap_curr[i] = 10 * MAX_N_NODES * k + i;
                    smallest_trees_curr[i] = 0;
                }
                
                // this inner loop is not parallelized
                for (longint local_tree_index = device_index + part_num*num_devices; local_tree_index < total_num_trees; local_tree_index = local_tree_index + num_threads_for_openmp*num_devices){
                    
                    generate_tree_from_index( num_tree_vertices, local_tree_index, local_permutation_array, local_degrees,local_tree, local_adjacency_mat , local_final_degrees, local_adjacency_list );
                    
                    realnumber cost = 0;
                    cost  = tree_prior(num_tree_vertices, local_adjacency_mat, local_adjacency_list, local_final_degrees); //here we include a prior cost on the tree  

                    if (cost < 10000){
                        switch (cost_choice) {
                            case '1':
                                cost += cost_trans_function(tree_cost_bei_array(num_tree_vertices, T, input_data, root_node, tree, local_adjacency_mat, local_final_degrees, local_adjacency_list));
                                break;
                            case '2':
                                cost += cost_trans_function(tree_cost_bei_array_inner(num_tree_vertices, T, input_data, root_node, tree, local_adjacency_mat, local_final_degrees, local_adjacency_list));
                                break;
                            case '3':
                                cost += cost_trans_function(tree_cost(num_tree_vertices, T, input_data, root_node, tree, local_adjacency_mat, local_final_degrees, local_adjacency_list));
                                break;
                            case '4':
                                cost += cost_trans_function(tree_cost_inner(num_tree_vertices, T, input_data, root_node, tree, local_adjacency_mat, local_final_degrees, local_adjacency_list));
                                break;
                            case '5':
                                for (int x = 0; x < n_reg_params; ++x) {
                                    // Call_to_path_algorithm(costs, ...);
                                } 
                                break;
                            default:
                                printf("Invalid cost choice!\n");
                                break;
                        }
                    }

                    if (cost_choice == '5') {
                        for (int x = 0; x < n_reg_params; ++x) {
                            heapify(costs[x], local_tree_index, list_of_maxheaps_foreach_regularization_param[x], list_of_smallest_trees_foreach_regularization_param[x], k);
                        }
                    }
                    else if (cost == cost && cost < max_heap_curr[0]) { // Avoid NaN
                        heapify(cost, local_tree_index, max_heap_curr, smallest_trees_curr, k);
                    }
                }
                
                // Write top k stuff to global memory:
                longint thread_id = omp_get_thread_num();
                if (cost_choice != '5') {
                    for (shortint i = 0; i < k; ++i) {
                        max_heap_mp[thread_id + i * num_threads_for_openmp] = max_heap_curr[i]; // part_num = thread index
                        smallest_trees_mp[thread_id + i * num_threads_for_openmp] = smallest_trees_curr[i];
                        //max_heap_mp[thread_id * k + i] = max_heap_curr[i]; // part_num = thread index
                        //smallest_trees_mp[thread_id * k + i] = smallest_trees_curr[i];
                    }
                } 
                // For case with solving full path and interpolating: 
                else { 
                    for (int x = 0; x < n_reg_params; ++x) {
                        memcpy(smallest_trees_with_regularization_mp[thread_id] + (x*k), list_of_smallest_trees_foreach_regularization_param[x], sizeof(longint) * k);
                        memcpy(max_heap_with_regularization_mp[thread_id] + (x*k), list_of_maxheaps_foreach_regularization_param[x], sizeof(realnumber) * k);
                    }
                }
            }
            realnumber end = omp_get_wtime();
            cpu_time = (float) 1000*(end - start);
        }
        
        printf("%f = CPU time\n", cpu_time);
        //fprintf(run_time, "%f\n", cpu_time);
        
        shortint stack[MAX_N_NODES];
        shortint visited[MAX_N_NODES];
        shortint father_list[2*num_tree_vertices + 1];
        for (shortint i = 0; i < num_tree_vertices; i++){
            visited[i] = 0;
        }
        
        if (device_choice == 'C'){
            printf("%d trees with smallest costs (tree index, cost): \n", k);
            for (shortint i = 0; i < k; ++i) {
                printf("(%lu, %f)\n", smallest_trees[i], max_heap[i]);
                generate_tree_from_index(num_tree_vertices, smallest_trees[i], permutation_array, degrees, tree, adjacency_mat, final_degrees, adjacency_list);
                dfs_tree_compute_fathers_non_recursive_array(num_tree_vertices, father_list, adjacency_list, final_degrees, root_node, stack, visited);
                fprintf(output_best_k, "%f\n", max_heap[i]);
                for (shortint j = 0 ; j < num_tree_vertices ; ++j){
                    if (j != root_node) {
                        printf("(%d, %d)\n", j + 1, father_list[j + 1]);
                        fprintf(output_best_k, "(%d, %d)\n", j + 1, father_list[j + 1]);
                    }
                }
                printf("\n");
                fprintf(output_best_k, "\n");
            }
            printf("\n");
        }
        else {
            // Merge all local top k from all threads together:
            for (int i = 0; i < k * num_threads_for_openmp; ++i) {
                heapify(max_heap_mp[i], smallest_trees_mp[i], max_heap, smallest_trees, k);
            }
            printf("%d trees with smallest costs (tree index, cost): \n", k);
            for (shortint i = 0; i < k; ++i) {
                printf("(%lu, %f)\n", smallest_trees[i], max_heap[i]);
                generate_tree_from_index(num_tree_vertices, smallest_trees[i], permutation_array, degrees, tree, adjacency_mat, final_degrees, adjacency_list);
                dfs_tree_compute_fathers_non_recursive_array(num_tree_vertices, father_list, adjacency_list, final_degrees, root_node, stack, visited);
                fprintf(output_best_k, "%f\n", max_heap[i]);
                for (shortint j = 0 ; j < num_tree_vertices ; ++j){
                    if (j != root_node) {
                        printf("(%d, %d)\n", j + 1, father_list[j + 1]);
                        fprintf(output_best_k, "(%d, %d)\n", j + 1, father_list[j + 1]);
                    }
                }
                printf("\n");
                fprintf(output_best_k, "\n");
            }
            printf("\n");
        }
    }
    //
    //
    // ***************************** START OF GPU CODE *****************************************************************************
    //
    //
    //
    else if (device_choice == 'G') {
        hipSetDevice(atoi(argv[8]));
        hipDeviceReset();
        
        int numthreadsperblock = 32;
        int numblocks =  128;

        if (argc == 14){
            numthreadsperblock = atoi(argv[12]);
            numblocks = atoi(argv[13]);
        }

        printf("Cuda is using %d threads per block and %d blocks\n",numthreadsperblock,numblocks);


        int chunck_per_cycle = numblocks*numthreadsperblock;
        //int num_trees_per_thread = ((total_num_trees + chunck_per_cycle - 1)/ chunck_per_cycle );
        
        // Allocate arrays for local top k trees for all CUDA threads:
        realnumber *max_heap_host = (realnumber *) malloc(topk * chunck_per_cycle * sizeof(realnumber));
        longint *smallest_trees_host = (longint *) malloc(topk * chunck_per_cycle * sizeof(longint));
        for (int i = 0; i < topk * chunck_per_cycle; ++i) {
            max_heap_host[i] = 100 * topk * chunck_per_cycle - i;
            smallest_trees_host[i] = 0;
        }
        
        // create space in the GPU global memory to store the best/top k values found
        realnumber *max_heap_device;
        longint *smallest_trees_device;
        hipMalloc((void **)&max_heap_device, topk * chunck_per_cycle * sizeof(realnumber));
        hipMalloc((void **)&smallest_trees_device, topk * chunck_per_cycle * sizeof(longint));
        
        // Initialize GPU global max_heap stuff to have heap property:
        hipMemcpy((void*) max_heap_device, (void*) max_heap_host, topk * chunck_per_cycle * sizeof(realnumber), hipMemcpyHostToDevice);
        hipMemcpy((void*) smallest_trees_device, (void*) smallest_trees_host, topk * chunck_per_cycle * sizeof(longint), hipMemcpyHostToDevice);
        
        // create space in the GPU to store the mutation frequency data
        realnumber * data_device;
        hipMalloc((void **)&data_device, T*num_tree_vertices*sizeof(realnumber) );
        hipMemcpy( (void*) data_device , (void*) input_data , T*num_tree_vertices*sizeof(realnumber) , hipMemcpyHostToDevice );
        
        // now we need to create space in the GPU to accomodate all the scrap memory we need
        
        int amount_of_mem_per_thread = sizeof(edge)*(num_tree_vertices-1) + sizeof(shortint)*(num_tree_vertices*num_tree_vertices + num_tree_vertices + (num_tree_vertices)*(num_tree_vertices) + num_tree_vertices-2 + num_tree_vertices); // + sizeof_treecost_scrapmem
        
        void * device_scrape_memory;
        hipMalloc((void **)&device_scrape_memory,amount_of_mem_per_thread*numblocks*numthreadsperblock ); // here we allocate all the scrape memory that we need
        
        gstart();
        
        switch (cost_choice) {
            case '1':
                kernel_to_compute_optimal_tree_only_global_mem_1k<<<numblocks,numthreadsperblock>>>(chunck_per_cycle, num_devices, device_index, total_num_trees,  num_tree_vertices, T, root_node, data_device, device_scrape_memory, amount_of_mem_per_thread, max_heap_device, smallest_trees_device, k);
                break;
            case '2':
                kernel_to_compute_optimal_tree_only_global_mem_2k<<<numblocks,numthreadsperblock>>>(chunck_per_cycle, num_devices, device_index, total_num_trees,  num_tree_vertices, T, root_node, data_device, device_scrape_memory, amount_of_mem_per_thread, max_heap_device, smallest_trees_device, k);
                break;
            case '3':
                kernel_to_compute_optimal_tree_only_global_mem_3k<<<numblocks,numthreadsperblock>>>(chunck_per_cycle, num_devices, device_index, total_num_trees,  num_tree_vertices, T, root_node, data_device, device_scrape_memory, amount_of_mem_per_thread, max_heap_device, smallest_trees_device, k);
                break;
            case '4':
                kernel_to_compute_optimal_tree_only_global_mem_4k<<<numblocks,numthreadsperblock>>>(chunck_per_cycle, num_devices, device_index, total_num_trees,  num_tree_vertices, T, root_node, data_device, device_scrape_memory, amount_of_mem_per_thread, max_heap_device, smallest_trees_device, k);
                break;
            default:
                printf("Invalid cost choice!\n");
                break;
        }
        
        gend(&gpu_time);
        printf("%f = GPU time\n",gpu_time);
        //fprintf(run_time, "%f\n", gpu_time);
        
        // Copy all local top k stuff back to CPU memory:
        hipMemcpy((void*) max_heap_host, (void*) max_heap_device, topk * chunck_per_cycle * sizeof(realnumber), hipMemcpyDeviceToHost);
        hipMemcpy((void*) smallest_trees_host, (void*) smallest_trees_device, topk * chunck_per_cycle * sizeof(longint), hipMemcpyDeviceToHost);
        
        shortint stack[MAX_N_NODES];
        shortint visited[MAX_N_NODES];
        shortint father_list[2*num_tree_vertices + 1];
        for (shortint i = 0; i < num_tree_vertices; i++){
            visited[i] = 0;
        }
        
        if (k <= 100 && k <= total_num_trees) {
            // Merge all local top k from all threads together:
            for (int i = 0; i < topk * chunck_per_cycle; ++i) {
                heapify(max_heap_host[i], smallest_trees_host[i], max_heap, smallest_trees, k);
            }
            printf("%d trees with smallest costs (tree index, cost): \n", k);
            for (shortint i = 0; i < k; ++i) {
                printf("(%lu, %f)\n", smallest_trees[i], max_heap[i]);
                generate_tree_from_index(num_tree_vertices, smallest_trees[i], permutation_array, degrees,tree, adjacency_mat, final_degrees, adjacency_list);
                dfs_tree_compute_fathers_non_recursive_array(num_tree_vertices, father_list, adjacency_list, final_degrees, root_node, stack, visited);
                fprintf(output_best_k, "%f\n", max_heap[i]);
                for (shortint j = 0 ; j < num_tree_vertices ; ++j){
                    if (j != root_node) {
                        printf("(%d, %d)\n", j + 1, father_list[j + 1]);
                        fprintf(output_best_k, "(%d, %d)\n", j + 1, father_list[j + 1]);
                    }
                }
                printf("\n");
                fprintf(output_best_k, "\n");
            }
            printf("\n");
        }
        else {
            printf("For GPU, k <= min(100, total number of trees)!\n");
            fprintf(output_best_k, "For GPU, k <= min(100, total number of trees)!\n");
        }
        
        gerror( hipPeekAtLastError() );
        hipDeviceSynchronize();
        
        hipFree(data_device);
        hipFree(max_heap_device);
        hipFree(smallest_trees_device);
        free(max_heap_host);
        free(smallest_trees_host);
    }
    //
    //
    // ***************************** END OF GPU CODE *****************************************************************************
    //
    //
    else {
        printf("Invalid choice of device!\n");
    }
    
    //free all the dynamically allocated memory in the CPU
    free(tree);
    free(degrees);
    free(permutation_array);
    //free(treecost_scrapmem);
    free(adjacency_list);
    free(adjacency_mat);
    free(final_degrees);
    free(max_heap_mp);
    free(smallest_trees_mp);

    free(regularization_parameters);
    
    // Close files:
    fclose(input_file);
    fclose(output_best_k);
    //fclose(run_time);
    
    
    return 0;
}
